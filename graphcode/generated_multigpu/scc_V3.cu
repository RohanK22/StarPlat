// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc_V3.h"

void Hong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int devicecount;
  hipGetDeviceCount(&devicecount);
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int** d_modified;
  d_modified = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(int)*(V+1));
  }

  int** d_scc;
  d_scc = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_scc[i], sizeof(int)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(int)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc[i],(int)-1);
  }
  bool fpoint1 = false; // asst in .cu 
  bool** h_fpoint1;
  h_fpoint1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i+=1){
    h_fpoint1[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint1;
  d_fpoint1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
  }


  int** d_modified_next;
  d_modified_next = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(int)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  int** h_modified;
  h_modified = (int**)malloc(sizeof(int*)*devicecount); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (int*)malloc(sizeof(int)*(V+1));
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Hong_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_fpoint1[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }





    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  int** d_visitFw;
  d_visitFw = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitFw[i], sizeof(int)*(V+1));
  }

  int** d_visitBw;
  d_visitBw = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitBw[i], sizeof(int)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw[i],(int)-1);
  }
  bool fpoint2 = false; // asst in .cu 
  bool** h_fpoint2;
  h_fpoint2 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i+=1){
    h_fpoint2[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint2;
  d_fpoint2 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint2[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  int** h_modified;
  h_modified = (int**)malloc(sizeof(int*)*devicecount); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (int*)malloc(sizeof(int)*(V+1));
  }

  while(!fpoint2) {

    fpoint2 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Hong_kernel2<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitBw[i],d_visitFw[i],d_fpoint2[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    bool fpoint3 = false; // asst in .cu 
    bool** h_fpoint3;
    h_fpoint3 = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i=0;i<devicecount;i+=1){
      h_fpoint3[i] = (bool*)malloc(sizeof(bool));
    }

    bool** d_fpoint3;
    d_fpoint3 = (bool**)malloc(sizeof(bool*)*devicecount);
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMalloc(&d_fpoint3[i],sizeof(bool));
      initKernel<bool> <<<1,1>>>(1,d_fpoint3[i],true);
    }


    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }

    int k=0; // #fixpt-Iterations
    int** h_modified;
    h_modified = (int**)malloc(sizeof(int*)*devicecount); 
    for (int i = 0 ; i < devicecount ; i++){
      h_modified[i] = (int*)malloc(sizeof(int)*(V+1));
    }

    while(!fpoint3) {

      fpoint3 = true;
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Hong_kernel3<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_visitFw[i],d_scc[i],d_fpoint3[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }



      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Hong_kernel4<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_visitBw[i],d_scc[i],d_fpoint3[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }



      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified[i],h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint3[i], d_fpoint3[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        fpoint3&=h_fpoint3[i][0];
      }
      k++;
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_fpoint3[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Hong_kernel5<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_visitFw[i],d_scc[i],d_visitBw[i],d_fpoint2[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    fpoint1 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }

    int k=0; // #fixpt-Iterations
    int** h_modified;
    h_modified = (int**)malloc(sizeof(int*)*devicecount); 
    for (int i = 0 ; i < devicecount ; i++){
      h_modified[i] = (int*)malloc(sizeof(int)*(V+1));
    }

    while(!fpoint1) {

      fpoint1 = true;
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        Hong_kernel6<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_fpoint1[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }





      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified[i],h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        fpoint1&=h_fpoint1[i][0];
      }
      k++;
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint2&=h_fpoint2[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
