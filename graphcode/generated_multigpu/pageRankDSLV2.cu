// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRankDSLV2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V+1;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  float** h_pageRank;
  h_pageRank = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_pageRank[i] = (float*)malloc(sizeof(float)*(V+1));
  }
  float** d_pageRank;
  d_pageRank = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank[i], sizeof(float)*(V+1));
  }


  //BEGIN DSL PARSING 
  float num_nodes = (float)g.num_nodes( ); // asst in .cu 
  float** h_num_nodes;
  h_num_nodes = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_num_nodes[i] = (float*)malloc(sizeof(float));
  }

  float** d_num_nodes;
  d_num_nodes = (float**)malloc(sizeof(float*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_num_nodes[i],sizeof(float));
    initKernel<float> <<<1,1>>>(1,d_num_nodes[i],g.num_nodes( ));
  }


  float** h_pageRank_nxt;
  h_pageRank_nxt = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_pageRank_nxt[i]=(float*)malloc(sizeof(float)*(V+1));
  }
  float** d_pageRank_nxt;
  d_pageRank_nxt = (float**)malloc(sizeof(float*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pageRank_nxt[i], sizeof(float)*(V+1));
  }

  float* h_pageRank_nxt_temp1 = (float*)malloc((V+1)*(devicecount)*sizeof(float));
  hipSetDevice(0);
  float* d_pageRank_nxt_temp1;
  hipMalloc(&d_pageRank_nxt_temp1,(V+1)*(devicecount)*sizeof(float));
  float* d_pageRank_nxt_temp2;
  hipMalloc(&d_pageRank_nxt_temp2,(V+1)*(devicecount)*sizeof(float));
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank[i],(float)1 / num_nodes);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpy(h_pageRank[i],d_pageRank[i],(V+1)*sizeof(float),hipMemcpyDeviceToHost);
  }
  int iterCount = 0; // asst in .cu 
  int** h_iterCount;
  h_iterCount = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_iterCount[i] = (int*)malloc(sizeof(int));
  }

  int** d_iterCount;
  d_iterCount = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_iterCount[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_iterCount[i],0);
  }


  float diff; // asst in .cu 
  float** h_diff;
  h_diff = (float**)malloc(sizeof(float*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_diff[i] = (float*)malloc(sizeof(float));
  }

  float** d_diff;
  d_diff = (float**)malloc(sizeof(float*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_diff[i],sizeof(float));
  }


  do{
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_PR_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_delta[i],d_num_nodes[i],d_pageRank[i],d_pageRank_nxt[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_pageRank_nxt_temp1+i*(V+1),d_pageRank_nxt[i],sizeof(float)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_pageRank_nxt_temp1,h_pageRank_nxt_temp1,(V+1)*(devicecount)*sizeof(float),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_pageRank_nxt_temp2+i*(V+1),h_pageRank_nxt[i],sizeof(float)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<float><<<numBlocks,threadsPerBlock>>>(d_pageRank_nxt_temp1,d_pageRank_nxt_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_pageRank_nxt[i],d_pageRank_nxt_temp1+i*(V+1),(V+1)*sizeof(float),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpy(d_pageRank_nxt[i],h_pageRank_nxt[i],(V+1)*sizeof(float),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipMemcpy(d_pageRank, d_pageRank_nxt, sizeof(float)*V, hipMemcpyDeviceToDevice);
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));
  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
