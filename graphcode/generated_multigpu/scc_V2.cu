// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc_V2.h"

void vHong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int devicecount;
  hipGetDeviceCount(&devicecount);
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  int** d_outDeg;
  d_outDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_outDeg[i], sizeof(int)*(V+1));
  }

  int** d_inDeg;
  d_inDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_inDeg[i], sizeof(int)*(V+1));
  }

  bool** d_visitFw;
  d_visitFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitFw[i], sizeof(bool)*(V+1));
  }

  bool** d_visitBw;
  d_visitBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitBw[i], sizeof(bool)*(V+1));
  }

  bool** d_propFw;
  d_propFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propFw[i], sizeof(bool)*(V+1));
  }

  bool** d_propBw;
  d_propBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propBw[i], sizeof(bool)*(V+1));
  }

  bool** d_isPivot;
  d_isPivot = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_isPivot[i], sizeof(bool)*(V+1));
  }

  int** d_scc;
  d_scc = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_scc[i], sizeof(int)*(V+1));
  }

  int** d_range;
  d_range = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_range[i], sizeof(int)*(V+1));
  }

  int** d_pivotField;
  d_pivotField = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pivotField[i], sizeof(int)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_outDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_inDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_isPivot[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_pivotField[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_outDeg[i],d_inDeg[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }




  bool fpoint1 = false; // asst in .cu 
  bool** h_fpoint1;
  h_fpoint1 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<devicecount;i+=1){
    h_fpoint1[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint1;
  d_fpoint1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
  }


  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel2<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }





    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel3<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_inDeg[i],d_pivotField[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }







  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel4<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }





  bool fpoint2 = false; // asst in .cu 
  bool** h_fpoint2;
  h_fpoint2 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<devicecount;i+=1){
    h_fpoint2[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint2;
  d_fpoint2 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint2[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!fpoint2) {

    fpoint2 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel5<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_propBw[i],d_visitBw[i],d_propFw[i],d_scc[i],d_visitFw[i],d_range[i],d_fpoint2[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }




    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint2&=h_fpoint2[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  fpoint1 = false;
  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel6<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }





    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel7<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }



  bool fpoint4 = false; // asst in .cu 
  bool** h_fpoint4;
  h_fpoint4 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<devicecount;i+=1){
    h_fpoint4[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint4;
  d_fpoint4 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint4[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],true);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!fpoint4) {

    fpoint4 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel8<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint4[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint4[i], d_fpoint4[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint4&=h_fpoint4[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  bool fpoint5 = false; // asst in .cu 
  bool** h_fpoint5;
  h_fpoint5 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<devicecount;i+=1){
    h_fpoint5[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint5;
  d_fpoint5 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint5[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],true);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!fpoint5) {

    fpoint5 = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel9<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_inDeg[i],d_pivotField[i],d_range[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }







    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel10<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }





    fpoint2 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }

    int k=0; // #fixpt-Iterations
    bool** h_modified;
    h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
    for (int i = 0 ; i < devicecount ; i++){
      h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
    }

    while(!fpoint2) {

      fpoint2 = true;
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        vHong_kernel11<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_propBw[i],d_visitBw[i],d_propFw[i],d_scc[i],d_visitFw[i],d_range[i],d_fpoint2[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }




      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      bool* d_modified_temp;
      bool* d_modified_temp1;
      hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
      hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
      initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
      hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        fpoint2&=h_fpoint2[i][0];
      }
      k++;
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel12<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitFw[i],d_visitBw[i],d_range[i],d_propBw[i],d_propFw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }





    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint5[i], d_fpoint5[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint5&=h_fpoint5[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
