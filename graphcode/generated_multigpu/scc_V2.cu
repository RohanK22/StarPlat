// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc_V2.h"

void vHong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V+1;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int** h_modified;
  h_modified = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_modified[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_modified;
  d_modified = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(int)*(V+1));
  }

  int* h_modified_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_modified_temp1;
  hipMalloc(&d_modified_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_modified_temp2;
  hipMalloc(&d_modified_temp2,(V+1)*(devicecount)*sizeof(int));
  int** h_outDeg;
  h_outDeg = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_outDeg[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_outDeg;
  d_outDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_outDeg[i], sizeof(int)*(V+1));
  }

  int* h_outDeg_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_outDeg_temp1;
  hipMalloc(&d_outDeg_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_outDeg_temp2;
  hipMalloc(&d_outDeg_temp2,(V+1)*(devicecount)*sizeof(int));
  int** h_inDeg;
  h_inDeg = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_inDeg[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_inDeg;
  d_inDeg = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_inDeg[i], sizeof(int)*(V+1));
  }

  int* h_inDeg_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_inDeg_temp1;
  hipMalloc(&d_inDeg_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_inDeg_temp2;
  hipMalloc(&d_inDeg_temp2,(V+1)*(devicecount)*sizeof(int));
  bool** h_visitFw;
  h_visitFw = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_visitFw[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_visitFw;
  d_visitFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitFw[i], sizeof(bool)*(V+1));
  }

  bool* h_visitFw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_visitFw_temp1;
  hipMalloc(&d_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_visitFw_temp2;
  hipMalloc(&d_visitFw_temp2,(V+1)*(devicecount)*sizeof(bool));
  bool** h_visitBw;
  h_visitBw = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_visitBw[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_visitBw;
  d_visitBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_visitBw[i], sizeof(bool)*(V+1));
  }

  bool* h_visitBw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_visitBw_temp1;
  hipMalloc(&d_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_visitBw_temp2;
  hipMalloc(&d_visitBw_temp2,(V+1)*(devicecount)*sizeof(bool));
  bool** h_propFw;
  h_propFw = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_propFw[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_propFw;
  d_propFw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propFw[i], sizeof(bool)*(V+1));
  }

  bool* h_propFw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_propFw_temp1;
  hipMalloc(&d_propFw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_propFw_temp2;
  hipMalloc(&d_propFw_temp2,(V+1)*(devicecount)*sizeof(bool));
  bool** h_propBw;
  h_propBw = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_propBw[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_propBw;
  d_propBw = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_propBw[i], sizeof(bool)*(V+1));
  }

  bool* h_propBw_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_propBw_temp1;
  hipMalloc(&d_propBw_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_propBw_temp2;
  hipMalloc(&d_propBw_temp2,(V+1)*(devicecount)*sizeof(bool));
  bool** h_isPivot;
  h_isPivot = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_isPivot[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_isPivot;
  d_isPivot = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_isPivot[i], sizeof(bool)*(V+1));
  }

  bool* h_isPivot_temp1 = (bool*)malloc((V+1)*(devicecount)*sizeof(bool));
  hipSetDevice(0);
  bool* d_isPivot_temp1;
  hipMalloc(&d_isPivot_temp1,(V+1)*(devicecount)*sizeof(bool));
  bool* d_isPivot_temp2;
  hipMalloc(&d_isPivot_temp2,(V+1)*(devicecount)*sizeof(bool));
  int** h_scc;
  h_scc = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_scc[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_scc;
  d_scc = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_scc[i], sizeof(int)*(V+1));
  }

  int* h_scc_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_scc_temp1;
  hipMalloc(&d_scc_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_scc_temp2;
  hipMalloc(&d_scc_temp2,(V+1)*(devicecount)*sizeof(int));
  int** h_range;
  h_range = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_range[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_range;
  d_range = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_range[i], sizeof(int)*(V+1));
  }

  int* h_range_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_range_temp1;
  hipMalloc(&d_range_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_range_temp2;
  hipMalloc(&d_range_temp2,(V+1)*(devicecount)*sizeof(int));
  int** h_pivotField;
  h_pivotField = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_pivotField[i]=(int*)malloc(sizeof(int)*(V+1));
  }
  int** d_pivotField;
  d_pivotField = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_pivotField[i], sizeof(int)*(V+1));
  }

  int* h_pivotField_temp1 = (int*)malloc((V+1)*(devicecount)*sizeof(int));
  hipSetDevice(0);
  int* d_pivotField_temp1;
  hipMalloc(&d_pivotField_temp1,(V+1)*(devicecount)*sizeof(int));
  int* d_pivotField_temp2;
  hipMalloc(&d_pivotField_temp2,(V+1)*(devicecount)*sizeof(int));
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(int)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified[i],d_modified[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_outDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_outDeg[i],d_outDeg[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_inDeg[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_inDeg[i],d_inDeg[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitFw[i],d_visitFw[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitBw[i],d_visitBw[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propFw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propFw[i],d_propFw[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propBw[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propBw[i],d_propBw[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_isPivot[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_isPivot[i],d_isPivot[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_scc[i],d_scc[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range[i],d_range[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_pivotField[i],(int)-1);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pivotField[i],d_pivotField[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_outDeg[i],d_inDeg[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_inDeg_temp1+i*(V+1),d_inDeg[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_inDeg_temp1,h_inDeg_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_inDeg_temp2+i*(V+1),h_inDeg[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_inDeg_temp1,d_inDeg_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_inDeg[i],d_inDeg_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_inDeg[i],h_inDeg[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_outDeg_temp1+i*(V+1),d_outDeg[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_outDeg_temp1,h_outDeg_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_outDeg_temp2+i*(V+1),h_outDeg[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_outDeg_temp1,d_outDeg_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_outDeg[i],d_outDeg_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_outDeg[i],h_outDeg[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  bool fpoint1 = false; // asst in .cu 
  bool** h_fpoint1;
  h_fpoint1 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint1[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint1;
  d_fpoint1 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],false);
  }


  int** d_modified_next;
  d_modified_next = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(int)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint1[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel2<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_scc_temp1+i*(V+1),d_scc[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_scc_temp1,h_scc_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_scc_temp2+i*(V+1),h_scc[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_scc_temp1,d_scc_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_scc[i],d_scc_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_scc[i],h_scc[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_isPivot_temp1+i*(V+1),d_isPivot[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_isPivot_temp1,h_isPivot_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_isPivot_temp2+i*(V+1),h_isPivot[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_isPivot_temp1,d_isPivot_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_isPivot[i],d_isPivot_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_isPivot[i],h_isPivot[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel3<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_pivotField[i],d_inDeg[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }




  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_pivotField_temp1+i*(V+1),d_pivotField[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_pivotField_temp1,h_pivotField_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_pivotField_temp2+i*(V+1),h_pivotField[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_pivotField_temp1,d_pivotField_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_pivotField[i],d_pivotField_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_pivotField[i],h_pivotField[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel4<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_isPivot_temp1+i*(V+1),d_isPivot[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_isPivot_temp1,h_isPivot_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_isPivot_temp2+i*(V+1),h_isPivot[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_isPivot_temp1,d_isPivot_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_isPivot[i],d_isPivot_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_isPivot[i],h_isPivot[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  bool fpoint2 = false; // asst in .cu 
  bool** h_fpoint2;
  h_fpoint2 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint2[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint2;
  d_fpoint2 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint2[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],false);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  while(!fpoint2) {

    fpoint2 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint2[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel5<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_propBw[i],d_visitBw[i],d_propFw[i],d_visitFw[i],d_scc[i],d_range[i],d_fpoint2[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propFw_temp1+i*(V+1),d_propFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_propFw_temp1,h_propFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_propFw_temp2+i*(V+1),h_propFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propFw_temp1,d_propFw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_propFw[i],d_propFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propFw[i],h_propFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propBw_temp1+i*(V+1),d_propBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_propBw_temp1,h_propBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_propBw_temp2+i*(V+1),h_propBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propBw_temp1,d_propBw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_propBw[i],d_propBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propBw[i],h_propBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint2&=h_fpoint2[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel6<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitFw[i],d_visitBw[i],d_range[i],d_propBw[i],d_propFw[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range_temp1+i*(V+1),d_range[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_range_temp1,h_range_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_range_temp2+i*(V+1),h_range[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_range_temp1,d_range_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_range[i],d_range_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_range[i],h_range[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propFw_temp1+i*(V+1),d_propFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_propFw_temp1,h_propFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_propFw_temp2+i*(V+1),h_propFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propFw_temp1,d_propFw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_propFw[i],d_propFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_propFw[i],h_propFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_propBw_temp1+i*(V+1),d_propBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_propBw_temp1,h_propBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_propBw_temp2+i*(V+1),h_propBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propBw_temp1,d_propBw_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_propBw[i],d_propBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_propBw[i],h_propBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_scc_temp1+i*(V+1),d_scc[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_scc_temp1,h_scc_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_scc_temp2+i*(V+1),h_scc[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_scc_temp1,d_scc_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_scc[i],d_scc_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_scc[i],h_scc[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  fpoint1 = false;
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    //printed here

    initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],(bool)false);
  }
  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  while(!fpoint1) {

    fpoint1 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint1[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel7<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint1[i],d_isPivot[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_scc_temp1+i*(V+1),d_scc[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_scc_temp1,h_scc_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_scc_temp2+i*(V+1),h_scc[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_scc_temp1,d_scc_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_scc[i],d_scc_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_scc[i],h_scc[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_isPivot_temp1+i*(V+1),d_isPivot[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_isPivot_temp1,h_isPivot_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_isPivot_temp2+i*(V+1),h_isPivot[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_isPivot_temp1,d_isPivot_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_isPivot[i],d_isPivot_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_isPivot[i],h_isPivot[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint1[i], d_fpoint1[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint1&=h_fpoint1[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint1[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range[i],(int)0);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range[i],d_range[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    vHong_kernel8<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i]);
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_range_temp1+i*(V+1),d_range[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  hipMemcpy(d_range_temp1,h_range_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(d_range_temp2+i*(V+1),h_range[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
  }
  Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_range_temp1,d_range_temp2,V,devicecount);
  for(int i=0;i<devicecount;i++){
    hipMemcpy(h_range[i],d_range_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(d_range[i],h_range[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  bool fpoint4 = false; // asst in .cu 
  bool** h_fpoint4;
  h_fpoint4 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint4[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint4;
  d_fpoint4 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint4[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],false);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  while(!fpoint4) {

    fpoint4 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint4[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel9<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_range[i],d_scc[i],d_fpoint4[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_range_temp1+i*(V+1),d_range[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_range_temp1,h_range_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_range_temp2+i*(V+1),h_range[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_range_temp1,d_range_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_range[i],d_range_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_range[i],h_range[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel10<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_range[i],d_fpoint4[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_range_temp1+i*(V+1),d_range[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_range_temp1,h_range_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_range_temp2+i*(V+1),h_range[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_range_temp1,d_range_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_range[i],d_range_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_range[i],h_range[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint4[i], d_fpoint4[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint4&=h_fpoint4[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint4[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  bool fpoint5 = false; // asst in .cu 
  bool** h_fpoint5;
  h_fpoint5 = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint5[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_fpoint5;
  d_fpoint5 = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint5[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],false);
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  while(!fpoint5) {

    fpoint5 = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_fpoint5[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel11<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_outDeg[i],d_pivotField[i],d_inDeg[i],d_range[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }




    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_pivotField_temp1+i*(V+1),d_pivotField[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_pivotField_temp1,h_pivotField_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_pivotField_temp2+i*(V+1),h_pivotField[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_pivotField_temp1,d_pivotField_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_pivotField[i],d_pivotField_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_pivotField[i],h_pivotField[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel12<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_pivotField[i],d_range[i],d_isPivot[i],d_visitBw[i],d_visitFw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }


    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_isPivot_temp1+i*(V+1),d_isPivot[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_isPivot_temp1,h_isPivot_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_isPivot_temp2+i*(V+1),h_isPivot[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_isPivot_temp1,d_isPivot_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_isPivot[i],d_isPivot_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_isPivot[i],h_isPivot[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    fpoint2 = false;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      //printed here

      initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],(bool)false);
    }
    // FIXED POINT variables
    //BEGIN FIXED POINT
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }

    while(!fpoint2) {

      fpoint2 = true;
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        initKernel<bool><<<1,1>>>(1,d_fpoint2[i],(bool)true);
      }
      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        vHong_kernel13<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_propBw[i],d_visitBw[i],d_propFw[i],d_visitFw[i],d_scc[i],d_range[i],d_fpoint2[i]);
      }

      for(int i=0;i<devicecount;i++)
      {
        hipSetDevice(i);
        hipDeviceSynchronize();
      }

      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propFw_temp1+i*(V+1),d_propFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_propFw_temp1,h_propFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_propFw_temp2+i*(V+1),h_propFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propFw_temp1,d_propFw_temp2,V,devicecount);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(h_propFw[i],d_propFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propFw[i],h_propFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_propBw_temp1+i*(V+1),d_propBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
      }
      hipSetDevice(0);
      hipMemcpy(d_propBw_temp1,h_propBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(d_propBw_temp2+i*(V+1),h_propBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
      }
      Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propBw_temp1,d_propBw_temp2,V,devicecount);
      for(int i=0;i<devicecount;i++){
        hipMemcpy(h_propBw[i],d_propBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_propBw[i],h_propBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_fpoint2[i], d_fpoint2[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
      }
      for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
      for(int i=0;i<devicecount;i++){
        fpoint2&=h_fpoint2[i][0];
      }
      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        initKernel<bool> <<<1,1>>>(1,d_fpoint2[i],true);
      }

      for(int i = 0 ; i < devicecount ; i++){
        hipSetDevice(i);
        hipDeviceSynchronize();
      }
    } // END FIXED POINT

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      vHong_kernel14<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_scc[i],d_visitFw[i],d_visitBw[i],d_range[i],d_propFw[i],d_propBw[i],d_fpoint5[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_range_temp1+i*(V+1),d_range[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_range_temp1,h_range_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_range_temp2+i*(V+1),h_range[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_range_temp1,d_range_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_range[i],d_range_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_range[i],h_range[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitFw_temp1+i*(V+1),d_visitFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitFw_temp1,h_visitFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitFw_temp2+i*(V+1),h_visitFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitFw_temp1,d_visitFw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitFw[i],d_visitFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitFw[i],h_visitFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_visitBw_temp1+i*(V+1),d_visitBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_visitBw_temp1,h_visitBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_visitBw_temp2+i*(V+1),h_visitBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_visitBw_temp1,d_visitBw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_visitBw[i],d_visitBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_visitBw[i],h_visitBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propFw_temp1+i*(V+1),d_propFw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_propFw_temp1,h_propFw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_propFw_temp2+i*(V+1),h_propFw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propFw_temp1,d_propFw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_propFw[i],d_propFw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propFw[i],h_propFw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_propBw_temp1+i*(V+1),d_propBw[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_propBw_temp1,h_propBw_temp1,(V+1)*(devicecount)*sizeof(bool),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_propBw_temp2+i*(V+1),h_propBw[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<bool><<<numBlocks,threadsPerBlock>>>(d_propBw_temp1,d_propBw_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_propBw[i],d_propBw_temp1+i*(V+1),(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_propBw[i],h_propBw[i],(V+1)*sizeof(bool),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_scc_temp1+i*(V+1),d_scc[i],sizeof(int)*(V+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    hipMemcpy(d_scc_temp1,h_scc_temp1,(V+1)*(devicecount)*sizeof(int),hipMemcpyHostToDevice);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_scc_temp2+i*(V+1),h_scc[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_correct<int><<<numBlocks,threadsPerBlock>>>(d_scc_temp1,d_scc_temp2,V,devicecount);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(h_scc[i],d_scc_temp1+i*(V+1),(V+1)*sizeof(int),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_scc[i],h_scc[i],(V+1)*sizeof(int),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_fpoint5[i], d_fpoint5[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      fpoint5&=h_fpoint5[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_fpoint5[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
