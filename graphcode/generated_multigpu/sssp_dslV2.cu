// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_dslV2.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int devicecount;
  hipGetDeviceCount(&devicecount);
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  int** h_dist;
  h_dist = (int**)malloc(sizeof(int*)*(devicecount));
  for(int i=0;i<devicecount;i++){
    h_dist[i] = (int*)malloc(sizeof(int)*(V+1));
  }
  int** d_dist;
  d_dist = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_dist[i], sizeof(int)*(V+1));
  }


  //BEGIN DSL PARSING 
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist[i],(int)INT_MAX);
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<bool><<<1,1>>>(V,d_modified[i],src,(bool)true); //InitIndexDevice
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<int><<<1,1>>>(V,d_dist[i],src,(int)0); //InitIndexDevice
  }
  bool finished = false; // asst in .cu 
  bool** h_finished;
  h_finished = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<devicecount;i+=1){
    h_finished[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_finished;
  d_finished = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_finished[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
  }


  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }

  int k=0; // #fixpt-Iterations
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1)); 
  for (int i = 0 ; i < devicecount ; i++){
    h_modified[i] = (bool*)malloc(sizeof(bool)*(V+1));
  }

  while(!finished) {

    finished = true;
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_SSSP_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_modified[i],d_modified_next[i],d_dist[i],d_finished[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }



    //ronaldo
    hipSetDevice(0);
    int* d_dist_temp;
    int* d_dist_temp1;
    hipMalloc(&d_dist_temp , (V+1)*sizeof(int));
    hipMalloc(&d_dist_temp1,(V+1)*(devicecount)*sizeof(int));
    initKernel<int><<<numBlocks,threadsPerBlock>>>(V+1,d_dist_temp,(int)INT_MAX);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_dist_temp1+i*(V+1),h_dist[i],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Min<<<numBlocks,numThreads>>>(d_dist_temp1,d_dist_temp,V,devicecount);
    hipMemcpy(h_dist[devicecount],d_dist_temp,sizeof(int)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_dist[i],h_dist[devicecount],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[i],d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    hipSetDevice(0);
    bool* d_modified_temp;
    bool* d_modified_temp1;
    hipMalloc(&d_modified_temp,(V+1)*sizeof(bool));
    hipMalloc(&d_modified_temp1,(devicecount)*(V+1)*sizeof(bool));
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V+1,d_modified_temp,false);
    for(int i=0;i<devicecount;i++){
      hipMemcpy(d_modified_temp1+i*(V+1),h_modified[i],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    Compute_Or<<<numBlocks,threadsPerBlock>>>(d_modified_temp1,d_modified_temp,V,devicecount);
    hipMemcpy(h_modified[devicecount],d_modified_temp,sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_finished[i], d_finished[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      finished&=h_finished[i][0];
    }
    k++;
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(dist,h_dist[devicecount], sizeof(int)*(V+1), hipMemcpyDeviceToHost);
} //end FUN
