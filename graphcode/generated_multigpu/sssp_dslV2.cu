// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_dslV2.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLARE DEVICE AND HOST vars in params
  int** h_dist;
  h_dist = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_dist[i] = (int*)malloc(sizeof(int)*(V+1));
  }
  int** d_dist;
  d_dist = (int**)malloc(sizeof(int*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_dist[i], sizeof(int)*(V+1));
  }


  //BEGIN DSL PARSING 
  bool** h_modified;
  h_modified = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i++){
    h_modified[i]=(bool*)malloc(sizeof(bool)*(V+1));
  }
  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified[i], sizeof(bool)*(V+1));
  }

  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist[i],(int)INT_MAX);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_dist[i],d_dist[i],(V+1)*sizeof(int),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++)
  {
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMemcpyAsync(h_modified[i],d_modified[i],(V+1)*sizeof(bool),hipMemcpyDeviceToHost);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  //hi2
  for(int i=0;i<devicecount;i++){
    h_modified[i][src]=true;
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<bool><<<1,1>>>(V,d_modified[i],src,(bool)true); //InitIndexDevice
  }
  //hi2
  for(int i=0;i<devicecount;i++){
    h_dist[i][src]=0;
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initIndex<int><<<1,1>>>(V,d_dist[i],src,(int)0); //InitIndexDevice
  }
  bool finished = false; // asst in .cu 
  bool** h_finished;
  h_finished = (bool**)malloc(sizeof(bool*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_finished[i] = (bool*)malloc(sizeof(bool));
  }

  bool** d_finished;
  d_finished = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_finished[i],sizeof(bool));
    initKernel<bool> <<<1,1>>>(1,d_finished[i],false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }


  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for (int i = 0; i < devicecount; i++) {
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i], sizeof(bool)*(V+1));
  }


  // FIXED POINT variables
  //BEGIN FIXED POINT
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  while(!finished) {

    finished = true;
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      initKernel<bool><<<1,1>>>(1,d_finished[i],(bool)true);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      Compute_SSSP_kernel1<<<numBlocks, threadsPerBlock>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_weight[i],d_src[i],d_rev_meta[i],d_dist[i],d_modified[i],d_modified_next[i],d_finished[i]);
    }

    for(int i=0;i<devicecount;i++)
    {
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    //global loop var v iden v
    //pull based

    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_dist[devicecount]+h_vertex_partition[i],d_dist[i]+h_vertex_partition[i],sizeof(int)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_dist[i],h_dist[devicecount],sizeof(int)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_modified[devicecount]+h_vertex_partition[i],d_modified_next[i]+h_vertex_partition[i],sizeof(bool)*(h_vertex_partition[i+1]-h_vertex_partition[i]),hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipMemcpyAsync(d_modified[i],h_modified[devicecount],sizeof(bool)*(V+1),hipMemcpyHostToDevice);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next[i], false);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipMemcpyAsync(h_finished[i], d_finished[i], sizeof(bool)*1, hipMemcpyDeviceToHost);
    }
    for(int i=0;i<devicecount;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
      finished&=h_finished[i][0];
    }
    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      initKernel<bool> <<<1,1>>>(1,d_finished[i],true);
    }

    for(int i = 0 ; i < devicecount ; i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
    }
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(    dist,   d_dist[0], sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN

int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  int *distance = (int *)malloc((g.num_nodes() + 1) * sizeof(int));
  int src = 1;
  Compute_SSSP(g, distance, src);
  for (int i = 0; i <= g.num_nodes(); i++)
  {
    std::cout << distance[i] << " ";
  }
  std::cout << std::endl;
  // std::cout << INT_MAX << std::endl;
  return 0;
}
