#include "PageRank_DSL_V2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));

  double* d_BC; hipMalloc(&d_BC, sizeof(double)*(V)); ///TODO from func

  //BEGIN DSL PARSING 
  float* d_num_nodes; hipMalloc(&d_num_nodes,sizeof(float)*(1));

  (float)initIndex<<<1,1>>>(1,d_num_nodes,0, 0);
  float* d_pageRank_nxt;
  hipMalloc(&d_pageRank_nxt, sizeof(float)*(V));

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,1 / num_nodes);

  int* d_iterCount; hipMalloc(&d_iterCount,sizeof(int)*(1));

  initIndex<<<1,1>>>(1,d_iterCount,0, 0);
  float* d_diff; hipMalloc(&d_diff,sizeof(float)*(1));

  initIndex<<<1,1>>>(1,d_diff,0, 0);
  do
  {diff = 0.000000;
    Compute_PR_kernel<<<numBlocks, numThreads>>>( V, E, d_meta, d_data, d_weight ,g,beta,delta,maxIter,
      d_pageRank);
    hipDeviceSynchronize();

    float* d_sum; hipMalloc(&d_sum,sizeof(float)*(1));

    float* d_val; hipMalloc(&d_val,sizeof(float)*(1));

    pageRank = pageRank_nxt;
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));//TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
  hipMemcpy(BC,d_BC , sizeof(double) * (V), hipMemcpyDeviceToHost);
} //end FUN
