#include "hip/hip_runtime.h"
#include"PageRank_DSL_V2.h"

__global__ void Compute_PR_kernel(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  float num_nodes = (float)g.num_nodes( );
  float* pageRank_nxt=new float[g.num_nodes()];
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    pageRank[t] = 1 / num_nodes;
  }
  int iterCount = 0;
  float diff = 0.0 ;
  do
  diff = 0.000000;
  pageRank = pageRank_nxt;
  iterCount++;
  while((diff > beta) && (iterCount < maxIter));
}