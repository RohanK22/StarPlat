// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRank_DSL_V2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }

  for(int i=0; i<= V; i++) {
    int temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  float num_nodes = (float)g.num_nodes( ); // asst in .cu

  float* pageRank_nxt = (float*) malloc(sizeof(float)*V);
  float* d_pageRank_nxt;
  hipMalloc(&d_pageRank_nxt, sizeof(float)*(V));

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,1 / num_nodes);

  int iterCount = 0; // asst in .cu

  float diff; // asst in .cu

  do
  {diff = 0.000000;
    hipMemcpyToSymbol(HIP_SYMBOL(::diff), &diff, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::num_nodes), &num_nodes, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::delta), &delta, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpy(d_pageRank, pageRank, sizeof(float)*(V), hipMemcpyHostToDevice);
    hipMemcpy(d_pageRank_nxt, pageRank_nxt, sizeof(float)*(V), hipMemcpyHostToDevice);
    Compute_PR_kernel<<<numBlocks, numThreads>>>(V,E,d_meta,d_data,d_weight,d_rev_meta,d_pageRank,d_pageRank_nxt);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&diff, HIP_SYMBOL(::diff), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&num_nodes, HIP_SYMBOL(::num_nodes), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&delta, HIP_SYMBOL(::delta), sizeof(float), 0, hipMemcpyDeviceToHost);
    hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
    hipMemcpy(pageRank_nxt, d_pageRank_nxt, sizeof(float)*(V), hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu

    for (int node = 0; node < V; node ++) 
    {
      pageRank [node] = pageRank_nxt [node] ;
    }
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));//TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
