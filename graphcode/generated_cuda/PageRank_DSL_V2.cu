// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "PageRank_DSL_V2.h"

void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }

  for(int i=0; i<= V; i++) {
    int temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;
  int* d_rev_meta;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;

  // For PageRank delta, beta and maxIter values
  float beta = 0.001;
  float delta = 0.85;
  int maxIter = 100;

  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  float* d_pageRank;
  hipMalloc(&d_pageRank, sizeof(float)*(V));


  //BEGIN DSL PARSING 
  float num_nodes(float) = g.num_nodes( ); // asst in .cu

  float* d_pageRank_nxt;
  hipMalloc(&d_pageRank_nxt, sizeof(float)*(V));

  initKernel<float> <<<numBlocks,threadsPerBlock>>>(V,d_pageRank,1 / num_nodes);

  int iterCount = 0; // asst in .cu

  float diff
  initIndex<<<1,1>>>(1,d_diff,0, 0);
  ; // asst in .cu

  float val
  initIndex<<<1,1>>>(1,d_val,0, 0);
  ; // asst in .cu

  do
  {diff = 0.000000;
    Compute_PR_kernel<<<numBlocks, numThreads>>>(V,E,d_meta,d_data,d_weight,g,beta,delta,maxIter,
      d_pageRank);
    ; // asst in .cu

    pageRank = pageRank_nxt;
    iterCount++;
  }while((diff > beta) && (iterCount < maxIter));//TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(pageRank, d_pageRank, sizeof(float)*(V), hipMemcpyDeviceToHost);
} //end FUN
