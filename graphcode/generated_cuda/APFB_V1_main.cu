// #include "PR_V1.h"
#include<bits/stdc++.h>
using namespace std;

#include "APFB_V1.cu"

//   int* h_rmatch = (int *)malloc((V)*sizeof(int));
//   int* h_cmatch = (int *)malloc((V)*sizeof(int));
//   hipMemcpy(h_rmatch, d_rmatch, V * sizeof(int), hipMemcpyDeviceToHost);
//   hipMemcpy(h_cmatch, d_cmatch, V * sizeof(int), hipMemcpyDeviceToHost);

//   int cntMatchings = 0;
//   for(int i = 0; i < V; i++){
//     // printf("rmatch[%d]: %d, cmatch[%d]: %d\n", i, h_rmatch[i], i, h_cmatch[i]);
//     if(h_cmatch[i] >= 0){
//       cntMatchings += 1;
//     }
//   }
//   printf("Maximum MATCHES: %d\n", cntMatchings);

int main(int argc, char** argv) {
    char* inp = argv[1];
    bool isWeighted = atoi(argv[2]) ? true : false;
    int nc = atoi(argv[3]);
    printf("Taking input from: %s\n", inp);
    
    graph g(inp);
    g.parseGraph(isWeighted);
    APFB(g, nc);
    return 0;
}