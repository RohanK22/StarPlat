// #include "PR_V1.h"
#include<bits/stdc++.h>
using namespace std;

#include "APFB_V1.cu"

int main(int argc, char** argv) {
    char* inp = argv[1];
    bool isWeighted = atoi(argv[2]) ? true : false;
    printf("Taking input from: %s\n", inp);
    
    int nc = 3;
    
//       bool* h_rmatch = (bool *)malloc((V)*sizeof(int));
//   bool* h_cmatch = (bool *)malloc((V)*sizeof(int));
//   hipMemcpy(h_rmatch, d_rmatch, V * sizeof(int), hipMemcpyDeviceToHost);
//   hipMemcpy(h_cmatch, d_rmatch, V * sizeof(int), hipMemcpyDeviceToHost);

//   for(int i = 0; i < V; i++){
//     printf("H rmatch[%d]: %d, cmatch[%d]: %d\n", i, h_rmatch[i], i, h_cmatch[i]);
//   }
    
    graph g(inp);
    g.parseGraph(isWeighted);
    APFB(g, nc);
    return 0;
}