// #include "PR_V1.h"
#include<bits/stdc++.h>
using namespace std;

#include "APFB_V1.cu"

//   int* h_rmatch = (int *)malloc((V)*sizeof(int));
//   int* h_cmatch = (int *)malloc((V)*sizeof(int));
//   hipMemcpy(h_rmatch, d_rmatch, V * sizeof(int), hipMemcpyDeviceToHost);
//   hipMemcpy(h_cmatch, d_cmatch, V * sizeof(int), hipMemcpyDeviceToHost);

//   for(int i = 0; i < V; i++){
//     printf("rmatch[%d]: %d, cmatch[%d]: %d\n", i, h_rmatch[i], i, h_cmatch[i]);
//   }

int main(int argc, char** argv) {
    char* inp = argv[1];
    bool isWeighted = atoi(argv[2]) ? true : false;
    printf("Taking input from: %s\n", inp);
    
    int nc = 6;
    
    graph g(inp);
    g.parseGraph(isWeighted);
    APFB(g, nc);
    return 0;
}