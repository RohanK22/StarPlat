#include "bc_dsl_v3.h"

void Compute_BC(graph& g,double* BC,std::set<int>& sourceSet)

{
  // CSR BEGIN
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? V: 512;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  //END CSR 

  //DECLAR DEVICE AND HOST vars in params
  double * d_BC;hipMalloc(&d_BC, sizeof(int)*(V));

  //BEGIN DSL PARSING 
  initKernel<double> <<<numBlocks,numThreads>>>(V,d_BC,0);

  double* d_sigma;
  hipMalloc(&d_sigma, sizeof(double)*(V));

  double* d_delta;
  hipMalloc(&d_delta, sizeof(double)*(V));

  //FOR SIGNATURE of SET
  std::set<int>::iterator itr;
  for(itr=sourceSet.begin();itr!=sourceSet.end();itr++)
  {
    unsigned src = (unsigned)*itr;
    initKernel<double> <<<numBlocks,numThreads>>>(V,d_delta,0);

    initKernel<double> <<<numBlocks,numThreads>>>(V,d_sigma,0);

    initIndex<double><<<1,1>>>(V,d_sigma,src,1);

    //EXTRA vars for ITBFS AND REVBFS
    bool finished;
    int hops_from_source=0;
    bool* d_finished;       hipMalloc(&d_finished,sizeof(bool) *(1));
    int* d_hops_from_source;hipMalloc(&d_hops_from_source, sizeof(int));
    int* d_level;           hipMalloc(&d_level,sizeof(int) *(V));

    initKernel<int> <<<numBlocks,numThreads>>>(V,d_level,-1);
    // long k =0 ;// For DEBUG
    do {
      finished = true;
      hipMemcpy(d_finished, &finished, sizeof(bool)*(1), hipMemcpyHostToDevice);

      //Kernel LAUNCH
      fwd_pass<<<numBlocks,numThreads>>>(V, d_meta, d_data,d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished, d_BC);

      incrementDeviceVar<<<1,1>>>(d_hops_from_source);
      hipDeviceSynchronize(); //MUST - rupesh
      ++hops_from_source; // updating the level to process in the next iteration
      // k++; //DEBUG

      hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    }while(!finished);

    hops_from_source--;
    hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);

    //BACKWARD PASS
    while(hops_from_source > 1) {

      //KERNEL Launch
      back_pass<<<numBlocks,numThreads>>>(V, d_meta, d_data, d_weight, d_delta, d_sigma, d_level, d_hops_from_source, d_finished, d_BC);

      hops_from_source--;
      hipMemcpy(d_hops_from_source, &hops_from_source, sizeof(int)*(1), hipMemcpyHostToDevice);
    }
  }
  //ADD TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

}