// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "SCC_V1.h"
#include<bits/stdc++.h>
using namespace std;


void Hong(graph& g)

{bool* d_isMSTEdge;
hipMalloc(&d_isMSTEdge, sizeof(bool)*(1));
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_modified;
  hipMalloc(&d_modified, sizeof(int)*(V));

  int* d_scc;
  hipMalloc(&d_scc, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(int)false);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc,(int)-1);

  bool fpoint1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_isMSTEdge);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT

  int* d_visitFw;
  hipMalloc(&d_visitFw, sizeof(int)*(V));

  int* d_visitBw;
  hipMalloc(&d_visitBw, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw,(int)-1);

  bool fpoint2 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!fpoint2) {

    fpoint2 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_visitBw,d_visitFw,d_isMSTEdge);
    hipDeviceSynchronize();



    bool fpoint3 = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!fpoint3) {

      fpoint3 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_visitFw,d_isMSTEdge);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);



      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_visitBw,d_isMSTEdge);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);




      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT

    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_visitBw,d_scc,d_visitFw,d_isMSTEdge);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    fpoint1 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!fpoint1) {

      fpoint1 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_isMSTEdge);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu


      hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT


    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT
int* h_scc;
    h_scc = (int *)malloc( (V)*sizeof(int));
    hipMemcpy(  h_scc,   d_scc, sizeof(int)*(V), hipMemcpyDeviceToHost);
    int scc_cnt = 0;
    for(int i = 0; i < V; i++) {

      if(h_scc[i] == i) {
        scc_cnt++;
      }
    }cout << endl;
    cout << "No.of strongly connected components in graph : " << scc_cnt << endl;


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_visitBw);
  hipFree(d_visitFw);
  hipFree(d_scc);
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
int main(int argc, char** argv) {
   time_t start, end;
    time(&start);
    char* inp = argv[1];
    bool isWeighted = atoi(argv[2]) ? true : false;
    printf("Taking input from: %s\n", inp);
    graph g(inp);
    g.parseGraph(isWeighted);
    Hong(g);
    time(&end);
    double time_taken = double(end - start);
    cout << "Time taken by program is : " << fixed
         << time_taken << setprecision(5);
    cout << " sec " << endl;
    return 0;
}
