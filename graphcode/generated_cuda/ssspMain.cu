#include "hip/hip_runtime.h"
/* v1.0
 * TODO: 1 kernel SSSP and CPU function - Thu, 15-Jul-2021, 12:21:20 IST
 * DONE: 1 kernel SSSP and CPU function - Thu, 15-Jul-2021, 22:24:12 IST
 * TODO: DUBUG LEVEL - Thu, 15-Jul-2021, 22:24:12 IST
 * DONE: DUBUG LEVEL - Thu, 15-Jul-2021, 23:01:02 IST
 * TODO: SEQ BF ALGO - Thu, 15-Jul-2021, 23:01:02 IST
 * DONE: SEQ BF ALGO - Thu, 15-Jul-2021, 23:24:02 IST
 * TODO: SEQ DIJKSTRA ALGO - Fri, 16-Jul-2021, 00:04:43 IST
 * DONE: SEQ DIJKSTRA ALGO - Fri, 16-Jul-2021, 00:22:34 IST
 *
 *
 * TODO
 * CLARIFY should wrap into Graph class?
 * CLARIFY should take file as param?
 * IMP should abstract for user
 * IMP should add timers
 */
#include "sssp_dsl.h"

#define DEBUG1 if(DEBUGCODE >= 1)
#define DEBUG2 if(DEBUGCODE >= 2)
unsigned DEBUGCODE = 0;

#define cudaCheckError() {                                             \
 hipError_t e=hipGetLastError();                                     \
 if(e!=hipSuccess) {                                                  \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0);                                                            \
 }                                                                     \
}

template <typename T>
__global__ void initKernel(unsigned nSize,T* dArray,T initVal){
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < nSize){
    dArray[id]=initVal;
  }
}

__global__ void SSSPKernel(unsigned nSize, int* csrMeta, int* csrData,int* csrWeight, int* distance, bool* modified){
  unsigned u = threadIdx.x + blockDim.x*blockIdx.x; // tid
  if(u < nSize && modified[u] ){ //Only modified u's && modified[u]
    for(int ii=csrMeta[u], end = csrMeta[u+1]; ii < end; ++ii){ //PUSH
      unsigned v = csrData[ii];
      int newDistance = distance[u] + csrWeight[ii] /*edgeweight(uv)*/ ;
      if(newDistance < distance[v]){
        atomicMin(&distance[v],newDistance);
        modified[v]=true;
      }
    }
  }
}

unsigned long long int processSSSP(const unsigned nSize, const unsigned mSize,
int*  csrM,
int*  csrD,
int*  csrW,
int*  distance,
bool* modified,
int source = 0)
{
  DEBUG1 std::cout<< "Process SSSP..start" << '\n';
  /**************
   * INITIALISE
   **************/
  // OPT we can do this in kernel
  for(int ii=0; ii< nSize; ++ii){
    distance[ii] = INT_MAX/2;
    modified[ii] = false;
  }

  distance[source] = 0    ;
  modified[source] = true ;

  /**********************
   * ALLOCATE on GPU
   **********************/
  int*  gcsrM;
  int*  gcsrD;
  int*  gcsrW;

  int*  gDistance;
  bool* gModified;

  unsigned nSizeIntPlus1  = sizeof(int) *(nSize+1);
  unsigned nSizeIntBytes  = sizeof(int) *( nSize );
  unsigned mSizeIntBytes  = sizeof(int) *( mSize );
  unsigned nSizeBoolBytes = sizeof(bool)*( nSize );

  hipMalloc(&gcsrM, nSizeIntPlus1);
  hipMalloc(&gcsrD, mSizeIntBytes);
  hipMalloc(&gcsrW, mSizeIntBytes);

  hipMalloc(&gDistance, nSizeIntBytes );
  hipMalloc(&gModified, nSizeBoolBytes);

  /***************
   * COPY TO GPU
   ***************/
  hipMemcpy( gcsrM, csrM, nSizeIntPlus1, hipMemcpyHostToDevice);
  hipMemcpy( gcsrD, csrD, mSizeIntBytes, hipMemcpyHostToDevice);
  hipMemcpy( gcsrW, csrW, mSizeIntBytes, hipMemcpyHostToDevice);

  hipMemcpy( gDistance, distance, nSizeIntBytes , hipMemcpyHostToDevice);
  hipMemcpy( gModified, modified, nSizeBoolBytes, hipMemcpyHostToDevice);

  cudaCheckError()

  /***************
   * LAUNCH CONFIG
   ***************/
  const unsigned numOfThreads = 1024;
  const unsigned numOfBlocks  = (nSize+numOfThreads-1)/numOfThreads; //Ceil(n/#threads) with function call
  bool finished = false;
  //~ hipSetDevice(1);
  DEBUG1 std::cout<< "FPLOCK..start.." << '\n';
  /***************
   * FIX PT COMPUTATION
   ***************/
   int k=0;
  do{

    SSSPKernel<<<numOfBlocks, numOfThreads>>>(nSize, gcsrM, gcsrD, gcsrW, gDistance, gModified);

    //~ hipDeviceSynchronize();

    thrust::device_ptr<bool> gModPtr = thrust::device_pointer_cast(gModified);
    finished = (thrust::reduce(gModPtr, gModPtr + nSize, true, thrust::logical_and<bool>())); // does this go into default stream?
    //~ finished = (thrust::reduce(gModPtr, gModPtr + nSize, false, thrust::logical_or<bool>()));
    //~ finished = !finished;
    /*
     * https://stackoverflow.com/questions/33156534/pass-cuda-array-to-thrustinclusive-scan/33156815#33156815
     * https://thrust.github.io/doc/functional_8h_source.html#l00798
     * finished = neg of all ORs   ~(a | b | c | 1) //not working
     * finished = all AND
     */

    hipDeviceSynchronize();

    //initKernel<bool> <<<numOfBlocks, numOfThreads>>>(nSize, gModified, false);
    //hipDeviceSynchronize();

    // DEBUG
    ++k;
    DEBUG2 printf("k:%d Finish:%s\n",k, (finished?"True":"False"));
    if(k==nSize){
      DEBUG1 std::cout<< "TERMINATED!";
      break;
    }
  }while(!finished);
  DEBUG1 std::cout<< "FPLOCK..END.." << '\n';

  hipMemcpy( distance, gDistance, nSizeIntBytes , hipMemcpyDeviceToHost);

  DEBUG1 std::cout<< "Process SSSP..Completed" << '\n';
  // TODO: TIMER END
  // TODO: Check with CPU

  unsigned long long int sum=0;

  for(int ii=0; ii < nSize; ++ii){
    DEBUG2 printf("d[%d]=%d\n",ii, distance[ii]);
    sum+=distance[ii];
  }
  //~ DEBUG1 printf("HASH:%lld\n",sum);

  cudaCheckError()

  hipFree(gcsrM);
  hipFree(gcsrD);
  hipFree(gcsrW);

  hipFree(gDistance);
  hipFree(gModified);

  return sum;
}



void readInput(
unsigned &nSize,
unsigned &mSize,
std::vector < std::vector <int>> &adjList,
std::map< std::pair<int,int>, int> &W){
  DEBUG1 std::cout<< "Reading inputs.." << '\n';

  std::string line, code, dummy;
  unsigned countEdge = 0;

  while(std::getline(std::cin, line) && !line.empty()){
    std::istringstream iStrSteam(line);
    iStrSteam >> code;
    if(code == "c") //Ignore this line
      continue;
    if(code == "p"){
      //~ p sp 264346 733846
      iStrSteam >> dummy >> nSize >> mSize;
      DEBUG2 std::cout<< nSize << " "<< mSize << '\n';
      adjList.resize(nSize);
    }
    else if( code == "a") {
      //~ a 1 2 803
      unsigned u, v;
      int w;

      iStrSteam >> u >> v >> w;
      //~ std::cout<< u << " " << v << " " << w << '\n';
      adjList[u-1].push_back(v-1);    // NOTE MINUS 1
      W[{u-1,v-1}]=w;                 // NOTE MINUS 1
      countEdge++;
    }
  }
  if(countEdge != mSize){
    std::cout<< "WARN: possibly different #edges" << '\n';
  }
  DEBUG1 std::cout<< "Loaded inputs.." << '\n';
}

void computeCSR(const unsigned nSize, const unsigned mSize,
const std::vector < std::vector <int>> &adjList,
const std::map< std::pair<int,int>, int> &W,  // DONT use const and LHS = W[]
int* csrM,
int* csrD,
int* csrW){
  DEBUG1 std::cout<< "Computing CSR.." << '\n';
  DEBUG2 std::cout<< "csr[0]=0" << '\n';
  for(int ii=0; ii < nSize; ++ii) {
    csrM[ii+1]= csrM[ii]+(int) adjList[ii].size() ; // NOTE +1
    DEBUG2 std::cout<< "csr"<<"["<<ii+1<< "]= "<< csrM[ii+1] << '\n';
  }

  int u   = 0;
  int idx = 0;
  DEBUG2 std::cout<< "CSRDATA" << '\n';
  for(auto nodes : adjList){
    // u begins with 0
    DEBUG2 std::cout<< u << ":" << ' ';
    for(int v : nodes){
      csrD[idx] = v ;
      csrW[idx] = W.at({u,v});
      DEBUG2 printf("(%d,%d),",csrD[idx],csrW[idx]);
      ++idx;
    }
    DEBUG2 std::cout<<  '\n';
    ++u; //NOTE
  }
  if( idx != mSize)
    std::cout<< "WARN: possibly different #edges" << '\n';

}

void printAdjList(const std::vector< std::vector<int> > &graph){
	int i = 0;
	for (auto vec : graph){

		std::cout << i << ": ";
		for(auto v : vec){
			std::cout<< v << " ";
		}
		i++;
		std::cout << std::endl;
	}
}

unsigned long long int BellmanFordMoore(
int nSize,
const std::vector< std::vector<int> > &adjList,
const std::map< std::pair<int,int>, int> &W,
int source = 0
) {
  DEBUG1 std::cout<< "SEQ CPU BFord..start" << '\n';
  std::vector<int> minDist(nSize, INT_MAX/2);
	//~ vector<int> parent(N, -1);
	// all parent are -1
	// all minDist are INT_MAX

	minDist[source] =0;
	bool updated = true;
	for (int k =1; k < (nSize-1) && updated; ++k){
		updated = false;
		DEBUG2 std::cout << k <<" of " << nSize-1 << std::endl;
		for(int u=0, endU = nSize; u < endU; u++){
			for(int j=0, endJ = adjList[u].size(); j < endJ; ++j){
				int v = adjList[u][j];
        int w = W.at({u,v}); // edge weight of (u,v)
				int newDist =  minDist[u]+w;

				if(	newDist < minDist[v] ){ // the to perform relax!
					minDist[v] = newDist;
					//~ parent[v] = u;
					updated=true;
				}
			}
		}
	}
  DEBUG1 std::cout<< "SEQ CPU BFord..end" << '\n';
  unsigned long long int sum = 0;
  for (auto &v : minDist)
    sum += v;

  return sum;
}


unsigned long long int Dijkstra( int nSize,
const std::vector< std::vector<int> > &adjList,
const std::map< std::pair<int,int>, int> &W,
int source=0
) {

  DEBUG1 std::cout<< "SEQ CPU Dijkstra..start" << '\n';

  std::vector<int> minDist(nSize, INT_MAX/2);
	//~ vector<int> parent(graph.size() , -1);
	minDist[ source ] = 0;
	std::set< std::pair<int,int> > active_vertices;
	active_vertices.insert( {0,source} );

	while (!active_vertices.empty()) {
		int u = active_vertices.begin()->second;

		active_vertices.erase( active_vertices.begin() );
		for (auto v : adjList[u]) {
			auto newdist = minDist[u] + W.at({u,v});
			if (newdist < minDist[v]) {
				active_vertices.erase( { minDist[v], v } );
				minDist[v] = newdist;
				//~ parent[v] = u;
				active_vertices.insert( { newdist, v } );
			}
		}
	}

  DEBUG1 std::cout<< "SEQ CPU Dijkstra..end" << '\n';
  unsigned long long int sum = 0;
  for (auto &v : minDist)
    sum += v;

  return sum;
}

int main(int argc, char** argv){

  if(argc > 0){
    DEBUGCODE = argc-1;
    DEBUG1 std::cout<< "DEBUG LEVEL:" << DEBUGCODE << '\n';
  }

  unsigned int nSize;
  unsigned int mSize;

  std::map< std::pair<int,int>, int> W;
  std::vector < std::vector <int>> adjList;

  readInput  (nSize, mSize, adjList, W);
  DEBUG2 printAdjList(adjList);

  int* csrM = (int*) malloc(sizeof(int)*(nSize+1)); //NOTE +1
  int* csrD = (int*) malloc(sizeof(int)*( mSize ));
  int* csrW = (int*) malloc(sizeof(int)*( mSize ));

  int*  distance = (int*)  malloc(sizeof(int) * nSize);
  bool* modified = (bool*) malloc(sizeof(bool)* nSize);
  //~ int*  parent= (int*) malloc(sizeof(int)*nSize);

  computeCSR (nSize, mSize, adjList, W, csrM, csrD, csrW);
  //~ initialize ()
  auto gpuHash = processSSSP(nSize, mSize, csrM, csrD, csrW, distance, modified);
  auto cpuHash = Dijkstra(nSize, adjList, W);
  //~ auto cpuHash = BellmanFordMoore(nSize, adjList, W);
  DEBUG1 printf("(G,C): (%lld,%lld)", gpuHash, cpuHash);
  printf(":%s\n", (gpuHash!=cpuHash?"False": "True"));
  //deallocate(csrM, csrD, csrW, distance, modified););
  free(csrM);
  free(csrD);
  free(csrW);
  free(distance);
  free(modified);

  return 0;
}
