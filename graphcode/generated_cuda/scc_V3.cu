// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "scc_V3.h"

void Hong(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_modified;
  hipMalloc(&d_modified, sizeof(int)*(V));

  int* d_scc;
  hipMalloc(&d_scc, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(int)false);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_scc,(int)-1);

  bool fpoint1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  int* d_visitFw;
  hipMalloc(&d_visitFw, sizeof(int)*(V));

  int* d_visitBw;
  hipMalloc(&d_visitBw, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw,(int)-1);

  bool fpoint2 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!fpoint2) {

    fpoint2 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_visitBw,d_visitFw);
    hipDeviceSynchronize();



    bool fpoint3 = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!fpoint3) {

      fpoint3 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_visitFw,d_scc);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);



      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint3), &fpoint3, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_visitBw,d_scc);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);




      hipMemcpyFromSymbol(&fpoint3, HIP_SYMBOL(::fpoint3), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc,d_visitBw,d_visitFw);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    fpoint1 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int k=0; // #fixpt-Iterations
    while(!fpoint1) {

      fpoint1 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
      Hong_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_scc);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu


      hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT


    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_visitFw);
  hipFree(d_visitBw);
  hipFree(d_scc);
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
