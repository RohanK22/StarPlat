#include "hip/hip_runtime.h"
#include"triangle_counting_DSL.h"

void Compute_TC(graph& g)

{
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  int* gpu_OA;
  int* gpu_edgeList;
  int* gpu_edgeList;

  hipMalloc(&gpu_OA, sizeof(int)*(1+V));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));

  if( V <= 1024)
  {
    block_size = V;
    num_blocks = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  hipMemcpy(&d_gpu_OA,OA, sizeof(int)*(1+V), hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  Compute_TC_kernel<<<num_blocks, block_size>>>(gpu_OA, gpu_edgeList, V, E ;
    hipDeviceSynchronize();


    long triangle_count = 0;

}