// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "MST_V2_Verified.h"
#include<bits/stdc++.h>
using namespace std;
void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;

  bool* d_isMSTEdge;
  hipMalloc(&d_isMSTEdge, sizeof(bool)*(E));

  initKernel<bool> <<<numBlocks_Edge,threadsPerBlock>>>(E,d_isMSTEdge,(bool)false);
  
 
  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  
  float milliseconds = 0;
  hipEventRecord(start1,0);

  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
   int* d_nodeId;
  hipMalloc(&d_nodeId, sizeof(int)*(V));

  int* d_color;
  hipMalloc(&d_color, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color,(int)-1);
  
  float time = 0;
  
  // hipEventRecord(start, 0);  
  Boruvka_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_nodeId,d_isMSTEdge);
  hipDeviceSynchronize();
  // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
  // printf("Boruvka_kernel_1 %.3f\n", time);

  ; // asst in .cu

  // hipEventRecord(start, 0);
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));
  
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  bool noNewComp = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
  // printf("d_modified malloc and init %.3f\n", time);
 
  while(!noNewComp) {
    // hipEventRecord(start, 0);
    noNewComp = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    int* d_minEdge;
    hipMalloc(&d_minEdge, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge,(int)-1);
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("d_minEdge malloc and init %.3f\n", time);


    // hipEventRecord(start, 0);
    Boruvka_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_color,d_isMSTEdge);
    hipDeviceSynchronize();
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Boruvka Kernel 2 %.3f\n", time);


    // hipEventRecord(start, 0);
    int* d_minEdgeOfComp;
    hipMalloc(&d_minEdgeOfComp, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp,(int)-1);
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("minEdgeOfComp malloc and init %.3f\n", time);

    // hipEventRecord(start, 0);
    bool finishedMinEdge = false; // asst in .cu
    
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!finishedMinEdge) {
      finishedMinEdge = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finishedMinEdge), &finishedMinEdge, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_minEdgeOfComp,d_color,d_isMSTEdge);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&finishedMinEdge, HIP_SYMBOL(::finishedMinEdge), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Boruvka Kernel 3 %.3f\n", time);

    // hipEventRecord(start, 0);
    Boruvka_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdgeOfComp,d_color,d_nodeId,d_isMSTEdge);
    hipDeviceSynchronize();
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Boruvka Kernel 4 %.3f\n", time);

    // hipEventRecord(start, 0);
    Boruvka_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdgeOfComp,d_color,d_nodeId,d_isMSTEdge);
    hipDeviceSynchronize();
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Boruvka Kernel 5 %.3f\n", time);

    // hipEventRecord(start, 0);
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    Boruvka_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdgeOfComp,d_color,d_nodeId,d_isMSTEdge);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Boruvka Kernel 6 %.3f\n", time);


    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!finished) {

      // hipEventRecord(start, 0);
      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      // printf("Single Iteration Symbol copy %.3f\n", time);     

      // hipEventRecord(start, 0);
      Boruvka_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_isMSTEdge);
      hipDeviceSynchronize();
      // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      // printf("Single Iteration Propagate colors %.3f\n", time);
     
      // hipEventRecord(start, 0);
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu

      ; // asst in .cu


      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
      // printf("Single Iteration Symbol copy %.3f\n", time);
  
    } // END FIXED POINT

    // hipEventRecord(start, 0);
    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_minEdgeOfComp);
    hipFree(d_minEdge);

    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    // hipEventRecord(stop, 0);  hipEventSynchronize(stop);  hipEventElapsedTime(&time, start, stop);
    // printf("Memcpy and initKernel %.3f\n", time);  
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop1,0); hipEventSynchronize(stop1); hipEventElapsedTime(&milliseconds, start1, stop1);
  printf("GPU Time: %.6f ms\n", milliseconds);

  bool* h_isMSTEdge = (bool *)malloc((E)*sizeof(bool));
  hipMemcpy(h_isMSTEdge, d_isMSTEdge, E * sizeof(bool), hipMemcpyDeviceToHost);

  long long mst = 0;
  for(int i = 0; i < E; i++){
    if(h_isMSTEdge[i] == true) mst += h_weight[i];
  }
  printf("MST Weight: %lld\n", mst);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_isMSTEdge);
  hipFree(d_color);
  hipFree(d_nodeId);
} //end FUN

int main(int argc, char** argv) {
  time_t start, end;
	time(&start);
  char* inp = argv[1];
  bool isWeighted = atoi(argv[2]) ? true : false;
  printf("Taking input from: %s\n", inp);
  graph g(inp);
  g.parseGraph(isWeighted);
  Boruvka(g);
	time(&end);
	double time_taken = double(end - start);
  cout << "Time taken by program is : " << fixed
        << time_taken << setprecision(5);
  cout << " sec " << endl;
  return 0;
}