// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "test_heap.h"

void Test(graph& g,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();







  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int maxSize = 100; // asst in .cu

  int siz = 4; // asst in .cu

  Heap hp;
  thrust::host_vector<int> cn;
  cn.resize(4);

  cn.push_back(2);

  cn.push_back(9);

  cn.push_back(1);

  cn.push_back(0);

  hp.insertE(cn,siz);

  thrust::host_vector<int> cnn;
  cnn = hp.deleteElem(2);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
