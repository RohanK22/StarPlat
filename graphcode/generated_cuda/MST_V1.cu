// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "MST_V1.h"

void Boruvka(graph& g)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  int* d_nodeId;
  hipMalloc(&d_nodeId, sizeof(int)*(V));

  int* d_color;
  hipMalloc(&d_color, sizeof(int)*(V));

  bool* d_isMSTEdge;
  hipMalloc(&d_isMSTEdge, sizeof(bool)*(E));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_nodeId,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_color,(int)-1);

  initKernel<bool> <<<numBlocks_Edge,threadsPerBlock>>>(E,d_isMSTEdge,(bool)false);

  Boruvka_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_nodeId,d_isMSTEdge);
  hipDeviceSynchronize();



  ; // asst in .cu

  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  bool noNewComp = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!noNewComp) {

    noNewComp = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
    int* d_minEdge;
    hipMalloc(&d_minEdge, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdge,(int)-1);

    Boruvka_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_color,d_isMSTEdge);
    hipDeviceSynchronize();



    int* d_minEdgeOfComp;
    hipMalloc(&d_minEdgeOfComp, sizeof(int)*(V));

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_minEdgeOfComp,(int)-1);

    Boruvka_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_minEdge,d_nodeId,d_color,d_minEdgeOfComp,d_isMSTEdge);
    hipDeviceSynchronize();



    Boruvka_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_nodeId,d_color,d_minEdgeOfComp,d_isMSTEdge);
    hipDeviceSynchronize();



    Boruvka_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_nodeId,d_color,d_minEdgeOfComp,d_isMSTEdge);
    hipDeviceSynchronize();



    bool finished = false; // asst in .cu

    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    while(!finished) {

      finished = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::noNewComp), &noNewComp, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
      Boruvka_kernel_6<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_nodeId,d_color,d_minEdgeOfComp,d_isMSTEdge);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);




      hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    } // END FIXED POINT

    Boruvka_kernel_7<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_color,d_isMSTEdge);
    hipDeviceSynchronize();



    ; // asst in .cu


    //hipFree up!! all propVars in this BLOCK!
    hipFree(d_minEdgeOfComp);
    hipFree(d_minEdge);

    hipMemcpyFromSymbol(&noNewComp, HIP_SYMBOL(::noNewComp), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified);
  hipFree(d_isMSTEdge);
  hipFree(d_color);
  hipFree(d_nodeId);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
  
  bool* h_isMSTEdge = (bool *)malloc( (E)*sizeof(bool));
  hipMemcpy(h_isMSTEdge, d_isMSTEdge, E * sizeof(bool), hipMemcpyDeviceToHost);

    int mst = 0;
  for(int i = 0; i < E; i++){
      if(h_isMSTEdge[i] == true) mst += h_weight[i];
  }
  printf("MST Weight: %d\n", mst);

} //end FUN

int main(int argc, char** argv) {
    char* inp = argv[1];
    bool isWeighted = atoi(argv[2]) ? true : false;
    printf("Taking input from: %s\n", inp);
    printf("Weighted: %d\n", isWeighted);
    graph g(inp);
    g.parseGraph(isWeighted);
    Boruvka(g);
    return 0;
}