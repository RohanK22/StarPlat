#include "sssp_dslV2.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,INT_MAX);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,false);

  initIndex<double><<<1,1>>>(V,d_modified,src,true); //InitIndexD
  initIndex<double><<<1,1>>>(V,d_dist,src,0); //InitIndexD
  bool finished = false; // asst in .cu

  // FIXED POINT variables
  bool* d_finished; hipMalloc(&d_finished,sizeof(bool)*(1));
  bool* d_modified; hipMalloc(&d_modified,sizeof(bool)*(V));
  //BEGIN FIXED POINT
  while(!finished) {
    initIndex<bool> <<< 1, 1>>>(1, d_finished,0, true);
    Compute_SSSP_kernel<<<numBlocks, numThreads>>>(V,E,d_meta,d_data,d_weight,g,d_dist,src);
    initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
    modified_nxt[v] = false ;
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(    dist,   d_dist, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
