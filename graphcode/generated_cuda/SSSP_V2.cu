// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "SSSP_V2.h"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));


  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,INT_MAX);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,false);

  initIndex<bool><<<1,1>>>(V,d_modified,src,(bool)true); //InitIndexDevice
  initIndex<int><<<1,1>>>(V,d_dist,src,(int)0); //InitIndexDevice
  bool finished = false; // asst in .cu

  // FIXED POINT variables
  bool* d_finished; hipMalloc(&d_finished,sizeof(bool)*(1));
  bool* d_modified_prev; hipMalloc(&d_modified_prev,sizeof(bool)*(V));
  bool* d_modified_next; hipMalloc(&d_modified_next,sizeof(bool)*(V));

  //BEGIN FIXED POINT
  int k=0; // #fixpt-Iterations
  while(!finished) {
    initIndex<bool> <<<1,1>>>(1,d_finished,0,true);
    Compute_SSSP_kernel<<<numBlocks, numThreads>>>(V,E,d_meta,d_data,d_weight,g,d_dist,src);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_prev, false);
    hipMemcpy(&finished, d_finished, sizeof(bool)*(1), hipMemcpyDeviceToHost);
    bool* tempModPtr = d_modified_next ; // SWAP next and prev ptrs
    d_modified_next = d_modified_prev;
    d_modified_prev = tempModPtr;
    k++;
  } // END FIXED POINT

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(    dist,   d_dist, sizeof(int)*(V), hipMemcpyDeviceToHost);
} //end FUN
