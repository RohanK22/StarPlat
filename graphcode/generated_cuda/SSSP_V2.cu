#include "SSSP_V2.h"
#include "../graph.hpp"

void Compute_SSSP(graph& g,int* dist,int src)

{
  // CSR BEGIN
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_weight;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_weight;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (numThreads+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  //END CSR 

  //DECLAR DEVICE AND HOST vars in params
  double* d_BC; hipMalloc(&d_BC, sizeof(double)*(V)); ///TODO from func

  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_dist,INT_MAX);

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,false);

  initIndex<double><<<1,1>>>(V,d_modified,src,true);
  initIndex<double><<<1,1>>>(V,d_dist,src,0);
  bool finished = false;
  while ( !finished[0] )
  {
    finished[0] = true;
    {
      Compute_SSSP_kernel<<<num_blocks, block_size>>>(gpu_OA, gpu_edgeList, V, E );
      hipDeviceSynchronize();

    }
     initKernel<bool> <<< 1, 1>>>(1, gpu_finished, true);
     Compute_SSSP_kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
     initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
     hipMemcpy(finished, gpu_finished,  sizeof(bool) *(1), hipMemcpyDeviceToHost);
    bool* tempModPtr = modified_nxt ;
    modified_nxt = modified_prev ;
    modified_prev = tempModPtr ;
    modified_nxt[v] = false ;
    //TIMER STOP
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %.6f ms\n", milliseconds);

    hipMemcpy(BC,d_BC , sizeof(double) * (V), hipMemcpyDeviceToHost);
  } //end FUN
