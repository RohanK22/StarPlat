#include "hip/hip_runtime.h"
#include"SSSP_V2.h"

__global__ void Compute_SSSP_kernel(graph& g,int* dist,int src)

{
  bool* modified=new bool[g.num_nodes()];
  bool* modified_nxt=new bool[g.num_nodes()];
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    dist[t] = INT_MAX;
    modified[t] = false;
    modified_nxt[t] = false;
  }
  modified[src] = true;
  dist[src] = 0;
  bool finished = false;
  while ( !finished[0] )
  {
    finished[0] = true;
    
    bool* tempModPtr = modified_nxt ;
    modified_nxt = modified_prev ;
    modified_prev = tempModPtr ;
    modified_nxt[v] = false ;

}