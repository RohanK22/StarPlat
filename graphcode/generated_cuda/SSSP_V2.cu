#include "hip/hip_runtime.h"
#include"SSSP_V2.h"

void Compute_SSSP(int * OA , int * edgeList , int* cpu_edgeLen  , int src)

{
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  int MAX_VAL = 2147483647 ;
  int * gpu_edgeList;
   int * gpu_edgeLen;
  int * gpu_dist;
   int * gpu_OA;
  bool * gpu_modified_prev;
  bool * gpu_finished;
  int *gpu_rev_OA;
  int *gpu_srcList;
  float  *gpu_node_pr;

  hipMalloc(&gpu_OA, sizeof(int)*(1+V));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));
  hipMalloc(&gpu_edgeLen, sizeof(int)*(E));
  hipMalloc(&gpu_dist, sizeof(int)*(V));
  hipMalloc(&gpu_modified_prev, sizeof(bool)*(V));
  hipMalloc(&gpu_modified_next, sizeof(bool)*(V));
  hipMalloc(&gpu_finished, sizeof(bool)*(1));
  hipMalloc(&gpu_srcList, sizeof(int)*(E));
  hipMalloc(&gpu_node_pr, sizeof(flaot)*(V));

  unsigned int block_size;
  unsigned int num_blocks;
  if( V <= 1024)
  {
    block_size = V;
    num_blocks = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  hipMemcpy(&d_gpu_OA,OA, sizeof(int)*(1+V), hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeLen,cpu_edgeLen , sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_dist,modified , sizeof(bool)*V, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_finished,finished , sizeof(bool)*1, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_srcList,cpu_srcList, sizeof(int)*(E), hipMemcpyHostToDevice);
  Compute_SSSP_kernel<<<num_blocks, block_size>>>(gpu_OA, gpu_edgeList, V, E ;
    hipDeviceSynchronize();


    bool* modified=new bool[g.num_nodes()];
    bool* modified_nxt=new bool[g.num_nodes()];
    for (int t = 0; t < V; g ++) 
    {
      dist[t] = INT_MAX;
      modified[t] = false;
      modified_nxt[t] = false;
    }
    modified[src] = true;
    dist[src] = 0;
    bool finished = false;
    while ( !finished[0] )
    {
      finished[0] = true;
      {
      }
       initKernel<bool> <<< 1, 1>>>(1, gpu_finished, true);
       Compute_SSSP_kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
       initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
       hipMemcpy(finished, gpu_finished,  sizeof(bool) *(1), hipMemcpyDeviceToHost);
      bool* tempModPtr = modified_nxt ;
      modified_nxt = modified_prev ;
      modified_prev = tempModPtr ;
      modified_nxt[v] = false ;

}