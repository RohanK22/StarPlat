// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "sssp_dslV3.h"

void Compute_SSSP(graph& g,int* dist,int* weight,int src
)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params
  int* d_dist;
  hipMalloc(&d_dist, sizeof(int)*(V));

  int* d_weight;
  hipMalloc(&d_weight, sizeof(int)*(E));


  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_nxt;
  hipMalloc(&d_modified_nxt, sizeof(bool)*(V));

  merged_kernel_1<<<numBlocks,threadsPerBlock>>>(V, d_dist, (int)INT_MAX, d_modified, (bool)false, d_modified_nxt, (bool)false, d_modified, src, (bool)true, d_dist, src, (int)0);
  bool finished = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!finished) {

    finished = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::finished), &finished, sizeof(bool), 0, hipMemcpyHostToDevice);
    Compute_SSSP_kernel<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_modified,d_weight,d_dist,d_modified_nxt);
    hipDeviceSynchronize();



    hipMemcpy(d_modified, d_modified_nxt, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified_nxt,(bool)false);


    hipMemcpyFromSymbol(&finished, HIP_SYMBOL(::finished), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT


  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_modified_nxt);
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

  hipMemcpy(    dist,   d_dist, sizeof(int)*(V), hipMemcpyDeviceToHost);
  hipMemcpy(  weight, d_weight, sizeof(int)*(E), hipMemcpyDeviceToHost);
} //end FUN
