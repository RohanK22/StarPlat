// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
#include "APFB_V1.h"

void APFB(graph& g,int nc)

{
  // CSR BEGIN
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  int *h_meta;
  int *h_data;
  int *h_src;
  int *h_weight;
  int *h_rev_meta;

  h_meta = (int *)malloc( (V+1)*sizeof(int));
  h_data = (int *)malloc( (E)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

  for(int i=0; i<= V; i++) {
    int temp = g.indexofNodes[i];
    h_meta[i] = temp;
    temp = g.rev_indexofNodes[i];
    h_rev_meta[i] = temp;
  }

  for(int i=0; i< E; i++) {
    int temp = g.edgeList[i];
    h_data[i] = temp;
    temp = g.srcList[i];
    h_src[i] = temp;
    temp = edgeLen[i];
    h_weight[i] = temp;
  }


  int* d_meta;
  int* d_data;
  int* d_src;
  int* d_weight;
  int* d_rev_meta;
  bool* d_modified_next;

  hipMalloc(&d_meta, sizeof(int)*(1+V));
  hipMalloc(&d_data, sizeof(int)*(E));
  hipMalloc(&d_src, sizeof(int)*(E));
  hipMalloc(&d_weight, sizeof(int)*(E));
  hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
  hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
  hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

  // CSR END
  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  unsigned numBlocks_Edge    = (E+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  int* d_rmatch;
  hipMalloc(&d_rmatch, sizeof(int)*(V));

  int* d_cmatch;
  hipMalloc(&d_cmatch, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_rmatch,(int)-1);

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_cmatch,(int)-1);

  bool noNewPaths = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  while(!noNewPaths) {

    noNewPaths = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::noNewPaths), &noNewPaths, sizeof(bool), 0, hipMemcpyHostToDevice);
    int L0 = 0; // asst in .cu

    int bfsLevel = L0; // asst in .cu

    int* d_bfsArray;
    hipMalloc(&d_bfsArray, sizeof(int)*(V));

    int NOT_VISITED = L0 - 1; // asst in .cu

    initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_bfsArray,(int)NOT_VISITED);

    hipMemcpyToSymbol(HIP_SYMBOL(::L0), &L0, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
    APFB_kernel_1<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_cmatch,d_bfsArray;
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&L0, HIP_SYMBOL(::L0), sizeof(int), 0, hipMemcpyDeviceToHost);
      hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);



      int* d_predeccesor;
      hipMalloc(&d_predeccesor, sizeof(int)*(V));

      initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_predeccesor,(int)-1);

      bool noNewVertices = false; // asst in .cu

      // FIXED POINT variables
      //BEGIN FIXED POINT
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      while(!noNewVertices) {

        noNewVertices = true;
        hipMemcpyToSymbol(HIP_SYMBOL(::noNewVertices), &noNewVertices, sizeof(bool), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(::NOT_VISITED), &NOT_VISITED, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(::bfsLevel), &bfsLevel, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(::noNewPaths), &noNewPaths, sizeof(bool), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(::noNewVertices), &noNewVertices, sizeof(bool), 0, hipMemcpyHostToDevice);
        APFB_kernel_2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_bfsArray,d_rmatch,d_predeccesor;
          hipDeviceSynchronize();
          hipMemcpyFromSymbol(&NOT_VISITED, HIP_SYMBOL(::NOT_VISITED), sizeof(int), 0, hipMemcpyDeviceToHost);
          hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);
          hipMemcpyFromSymbol(&bfsLevel, HIP_SYMBOL(::bfsLevel), sizeof(int), 0, hipMemcpyDeviceToHost);
          hipMemcpyFromSymbol(&noNewPaths, HIP_SYMBOL(::noNewPaths), sizeof(bool), 0, hipMemcpyDeviceToHost);
          hipMemcpyFromSymbol(&noNewVertices, HIP_SYMBOL(::noNewVertices), sizeof(bool), 0, hipMemcpyDeviceToHost);



          ; // asst in .cu

          bfsLevel = bfsLevel + 1;

          hipMemcpyFromSymbol(&noNewVertices, HIP_SYMBOL(::noNewVertices), sizeof(bool), 0, hipMemcpyDeviceToHost);
          hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
          initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
        } // END FIXED POINT

        bool* d_compress;
        hipMalloc(&d_compress, sizeof(bool)*(V));

        initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_compress,(bool)false);

        hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
        APFB_kernel_3<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_rmatch,d_compress;
          hipDeviceSynchronize();
          hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);



          bool compressed = false; // asst in .cu

          // FIXED POINT variables
          //BEGIN FIXED POINT
          initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
          while(!compressed) {

            compressed = true;
            hipMemcpyToSymbol(HIP_SYMBOL(::compressed), &compressed, sizeof(bool), 0, hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(::compressed), &compressed, sizeof(bool), 0, hipMemcpyHostToDevice);
            APFB_kernel_4<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_predeccesor,d_cmatch,d_compress,d_rmatch;
              hipDeviceSynchronize();
              hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);
              hipMemcpyFromSymbol(&compressed, HIP_SYMBOL(::compressed), sizeof(bool), 0, hipMemcpyDeviceToHost);



              ; // asst in .cu

              ; // asst in .cu


              hipMemcpyFromSymbol(&compressed, HIP_SYMBOL(::compressed), sizeof(bool), 0, hipMemcpyDeviceToHost);
              hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
              initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
            } // END FIXED POINT

            hipMemcpyToSymbol(HIP_SYMBOL(::nc), &nc, sizeof(int), 0, hipMemcpyHostToDevice);
            APFB_kernel_5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_weight,d_rev_meta,d_modified_next,d_rmatch,d_cmatch;
              hipDeviceSynchronize();
              hipMemcpyFromSymbol(&nc, HIP_SYMBOL(::nc), sizeof(int), 0, hipMemcpyDeviceToHost);



              ; // asst in .cu


              //hipFree up!! all propVars in this BLOCK!
              hipFree(d_compress);
              hipFree(d_predeccesor);
              hipFree(d_bfsArray);

              hipMemcpyFromSymbol(&noNewPaths, HIP_SYMBOL(::noNewPaths), sizeof(bool), 0, hipMemcpyDeviceToHost);
              hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
              initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
            } // END FIXED POINT


            //hipFree up!! all propVars in this BLOCK!
            hipFree(d_cmatch);
            hipFree(d_rmatch);
            hipFree(d_modified);

            //TIMER STOP
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("GPU Time: %.6f ms\n", milliseconds);

          } //end FUN
