#include "hip/hip_runtime.h"
#include"bc_dsl_v2.h"

void Compute_BC(graph& g,float* BC,std::set<int>& sourceSet)

{
  unsigned V = g.num_nodes();
  unsigned E = g.num_edges();

  int MAX_VAL = 2147483647 ;
  int * gpu_edgeList;
   int * gpu_edgeLen;
  int * gpu_dist;
   int * gpu_OA;
  bool * gpu_modified_prev;
  bool * gpu_finished;
  int *gpu_rev_OA;
  int *gpu_srcList;
  float  *gpu_node_pr;

  hipMalloc(&gpu_OA, sizeof(int)*(1+V));
  hipMalloc(&gpu_edgeList, sizeof(int)*(E));
  hipMalloc(&gpu_edgeLen, sizeof(int)*(E));
  hipMalloc(&gpu_dist, sizeof(int)*(V));
  hipMalloc(&gpu_modified_prev, sizeof(bool)*(V));
  hipMalloc(&gpu_modified_next, sizeof(bool)*(V));
  hipMalloc(&gpu_finished, sizeof(bool)*(1));
  hipMalloc(&gpu_srcList, sizeof(int)*(E));
  hipMalloc(&gpu_node_pr, sizeof(flaot)*(V));

  unsigned int block_size;
  unsigned int num_blocks;
  if( V <= 1024)
  {
    block_size = V;
    num_blocks = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  hipMemcpy(&d_gpu_OA,OA, sizeof(int)*(1+V), hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeList,edgeList, sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_edgeLen,cpu_edgeLen , sizeof(int)*E, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_dist,modified , sizeof(bool)*V, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_finished,finished , sizeof(bool)*1, hipMemcpyHostToDevice);
  hipMemcpy(&d_gpu_srcList,cpu_srcList, sizeof(int)*(E), hipMemcpyHostToDevice);
  Compute_BC_kernel<<<num_blocks, block_size>>>(gpu_OA, gpu_edgeList, V, E ;
    hipDeviceSynchronize();


    for (int t = 0; t < V; g ++) 
    {
      BC[t] = 0;
    }
    {
      int src = *itr;
      double* sigma=new double[g.num_nodes()];
      int* bfsDist=new int[g.num_nodes()];
      float* delta=new float[g.num_nodes()];
      for (int t = 0; t < V; g ++) 
      {
        delta[t] = 0;
        bfsDist[t] = -1;
      }
      for (int t = 0; t < V; g ++) 
      {
        sigma[t] = 0;
      }
      bfsDist[src] = 0;
      sigma[src] = 1;
      std::vector<std::vector<int>> levelNodes(g.num_nodes()) ;
      std::vector<std::vector<int>>  levelNodes_later(omp_get_max_threads()) ;
      std::vector<int>  levelCount(g.num_nodes()) ;
      int phase = 0 ;
      levelNodes[phase].push_back(src) ;
      int bfsCount = 1 ;
      levelCount[phase] = bfsCount;
      while ( bfsCount > 0 )
      {
         int prev_count = bfsCount ;
        bfsCount = 0 ;
        #pragma omp parallel for
        for( int l = 0; l < prev_count ; l++)
        {
          int v = levelNodes[phase][l] ;
          for(int edge = g.indexofNodes[v] ; edge < g.indexofNodes[v+1] ; edge++) {
            int nbr = g.edgeList[edge] ;
            int dnbr ;
            if(bfsDist[nbr]<0)
            {
              dnbr = __sync_val_compare_and_swap(&bfsDist[nbr],-1,bfsDist[v]+1);
              if (dnbr < 0)
              {
                int num_thread = omp_get_thread_num();
                 levelNodes_later[num_thread].push_back(nbr) ;
              }
            }
          }
          if(bfsDist[w]==bfsDist[v]+1)
          {
          }
        }
      }
      phase = phase + 1 ;
      for(int i = 0;i < omp_get_max_threads();i++)
      {
         levelNodes[phase].insert(levelNodes[phase].end(),levelNodes_later[i].begin(),levelNodes_later[i].end());
         bfsCount=bfsCount+levelNodes_later[i].size();
         levelNodes_later[i].clear();
      }
       levelCount[phase] = bfsCount ;
    }
    phase = phase -1 ;
    while (phase > 0)
    {
      #pragma omp parallel for
      for( int l = 0; l < levelCount[phase] ; l++)
      {
        int v = levelNodes[phase][l] ;
        if(bfsDist[w]==bfsDist[v]+1)
        {
        }
      }
      BC[v] = BC[v] + delta[v];
    }
    phase = phase - 1 ;
  }
}

}