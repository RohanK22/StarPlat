#include "hip/hip_runtime.h"
#include"PageRank_DSL.h"

__global__ void Compute_PR_kernel(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  float num_nodes = (float)g.num_nodes( );
  #pragma omp parallel for
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    pageRank[t] = 1 / num_nodes;
  }
  int iterCount = 0;
  float diff = 0.0 ;
  do
  {
    diff = 0.000000;
    unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int v =id
    {
      float sum = 0.000000;
      for (int edge = gpu_rev_OA[v]; edge < gpu_rev_OA[v+1]; edge ++) 
      {int nbr = srcList[edge] ;
        sum = sum + pageRank[nbr] / (gpu_OA[nbr+1]-gpu_OA[nbr]);
      }
      float val = (1 - delta) / num_nodes + delta * sum;
      diff = diff+ val - pageRank[v];
      pageRank[v] = val;
    }
    iterCount++;
  }
  while((diff > beta) && (iterCount < maxIter));
}