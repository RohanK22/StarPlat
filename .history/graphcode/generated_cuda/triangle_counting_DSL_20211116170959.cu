#include "hip/hip_runtime.h"
#include"triangle_counting_DSL.h"

__global__ void Compute_TC_kernel(graph& g)

{
  long triangle_count = 0;
  unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
  {
    for (int edge = g.indexofNodes[v]; edge < g.indexofNodes[v+1]; edge ++) 
    {int u = g.edgeList[edge] ;
      if (u < v ){
        for (int edge = g.indexofNodes[v]; edge < g.indexofNodes[v+1]; edge ++) 
        {int w = g.edgeList[edge] ;
          if (w > v ){
            if (g.check_if_nbr(u, w) ){
              triangle_count = triangle_count+ 1;
            }
          }
        }
      }
    }
  }

}