#include "hip/hip_runtime.h"
#include"sssp_dsl.h"

template <typename T>
__global__ void initKernel(unsigned V, T* init_array, T initVal)
{
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < V)
  {
    init_array[id]=initVal;
  }
}
__global__ void Compute_SSSP_kernel(graph g,int src)
{
  unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
  unsigned int v = id;
  if (id < V) 
  {
    if (gpu_modified_prev[id] ){
      for (int edge = gpu_OA[id]; edge < gpu_OA[id+1]; edge ++) 
      {
        int nbr = gpu_edgeList[edge] ;
        int e = edge;
         int dist_new = gpu_dist[v] + gpu_weight[e];
        if (gpu_dist[id] != MAX_VAL && gpu_dist[nbr] > dist_new)
        {
          atomicMin(&gpu_dist[nbr] , dist_new);
          gpu_modified_next[nbr]=true;
          gpu_finished[0] = false;
        }
      }
    }
  }

}
void SSSP(int * OA , int * edgeList , int* cpu_edgeLen  , int src ,int V, int E )
{
  int MAX_VAL = 2147483647 ;
  int * gpu_edgeList;
  int * gpu_edgeLen;
  int * gpu_dist;
  int * gpu_OA;
  bool * gpu_modified_prev;
  bool * gpu_modified_next;
  bool * gpu_finished;


  hipMalloc(&gpu_OA,sizeof(int) *(1+V));
  hipMalloc(&gpu_edgeList,sizeof(int) *(E));
  hipMalloc(&gpu_edgeLen,sizeof(int) *(E));
  hipMalloc(&gpu_dist,sizeof(int) *(V));
  hipMalloc(&gpu_modified_prev,sizeof(bool) *(V));
  hipMalloc(&gpu_modified_next,sizeof(bool) *(V));
  hipMalloc(&gpu_finished,sizeof(bool) *(1));

  unsigned int block_size;
  unsigned int num_blocks;
   if(V <= 1024)
   {
    block_size  = V;
    block_size  = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  bool * finished = new bool[1];
  finished[0] = false;
  int* dist=new int[V];
  bool* modified=new bool[V];
  for (int t = 0; t < V; t ++) 
  {
    dist[t] = INT_MAX;
    modified[t] = false;
  }
  modified[src] = true;
  dist[src] = 0;
  hipMemcpy (gpu_OA, OA, sizeof(int) *(1+V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeList, edgeList, sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeLen, cpu_edgeLen , sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_dist, dist, sizeof(int) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_modified_prev, modified , sizeof(bool) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_finished, finished , sizeof(bool) *(1) ,hipMemcpyHostToDevice);

  int k =0;

  while ( !finished[0] )
  {
    finished[0] =true;
    initKernel<bool><<<1,1>>>(1, gpu_finished, true);
    Compute_SSSP_kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
    hipDeviceSynchronize();
    initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
    hipDeviceSynchronize();
    bool *tempModPtr  = gpu_modified_next;
    gpu_modified_next = gpu_modified_prev;
    gpu_modified_prev = tempModPtr;
    ++k;
    if(k==V)
    {
      break;
    }
  }

  hipMemcpy(dist,gpu_dist , sizeof(int) * (V), hipMemcpyDeviceToHost);
  char *outputfilename = "output.txt";
  FILE *outputfilepointer;
  outputfilepointer = fopen(outputfilename, "w");
  for (int i = 0; i <V; i++)
  {
    fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  }

}
