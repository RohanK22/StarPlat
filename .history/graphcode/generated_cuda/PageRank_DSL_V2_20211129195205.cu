#include "hip/hip_runtime.h"
#include"PageRank_DSL_V2.h"

__global__ void Compute_PR_kernel(graph& g,float beta,float delta,int maxIter,
  float* pageRank)

{
  float num_nodes = (float)g.num_nodes( );
  float* pageRank_nxt=new float[g.num_nodes()];
  #pragma omp parallel for
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    pageRank[t] = 1 / num_nodes;
  }
  int iterCount = 0;
  float diff = 0.0 ;
  do
  {
    diff = 0.000000;
    unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int v =id
    {
      float sum = 0.000000;
      for (int edge = gpu_rev_OA[v]; edge < gpu_rev_OA[v+1]; edge ++) 
      {int nbr = srcList[edge] ;
        sum = sum + pageRank[nbr] / (gpu_OA[nbr+1]-gpu_OA[nbr]);
      }
      float val = (1 - delta) / num_nodes + delta * sum;
      diff = diff+ val - pageRank[v];
      pageRank_nxt[v] = val;
    }
    pageRank = pageRank_nxt;
    iterCount++;
  }
  while((diff > beta) && (iterCount < maxIter));
}