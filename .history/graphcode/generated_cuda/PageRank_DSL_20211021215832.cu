#include "hip/hip_runtime.h"
#include"PageRank_DSL.h"

__global__ void Compute_PR(graph& g,float beta,float delta,int maxIter,
  float* pageRank)
{
  float num_nodes = (float)g.num_nodes( );
  #pragma omp parallel for
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    pageRank[t] = 1 / num_nodes;
  }
  int iterCount = 0;
  float diff = 0.0 ;
  do
  {
    diff = 0.000000;
    .
    {
      float sum = 0.000000;
      for (int edge = g.rev_indexofNodes[v]; edge < g.rev_indexofNodes[v+1]; edge ++) 
      {int nbr = g.srcList[edge] ;
        sum = sum + pageRank[nbr] / (g.indexofNodes[nbr+1]-g.indexofNodes[nbr]);
      }
      float val = (1 - delta) / num_nodes + delta * sum;
      diff = diff+ val - pageRank[v];
      pageRank[v] = val;
    }
    iterCount++;
  }
  while((diff > beta) && (iterCount < maxIter));
}
void Compute_PR(int * rev_OA, int * OA, int * cpu_srcList , float * node_pr , int V, int E)
{
  int *gpu_rev_OA;
  int *gpu_srcList;
  int * gpu_OA;
  float * gpu_node_pr;
  hipMalloc( &gpu_rev_OA, sizeof(int) * (1+V) );
  hipMalloc( &gpu_OA, sizeof(int) * (1+V) );
  hipMalloc( &gpu_srcList, sizeof(int) * (E) );
  hipMalloc( &gpu_node_pr, sizeof(float) * (V) );
  unsigned int block_size;
  unsigned int num_blocks;
  for(int i=0; i< V; i++)
  {
    node_pr[i]= 1.0/V;
  }
  if(V <= 1024)
  {
    block_size = V;
    num_blocks = 1;
    else
    {
      block_size = 1024;
      num_blocks = ceil(((float)V) / block_size);
    }
    hipMemcpy(gpu_rev_OA, rev_OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
    hipMemcpy(gpu_OA, OA, sizeof(int) * (1+V), hipMemcpyHostToDevice);
    hipMemcpy(gpu_srcList, cpu_srcList, sizeof(int) * (E), hipMemcpyHostToDevice);
    hipMemcpy(gpu_node_pr, node_pr , sizeof(float) * (V), hipMemcpyHostToDevice);
    float beta = 0.001;
    float delta = 0.85;
    int maxIter = 100;
    int iterCount=0;
    float diff;
