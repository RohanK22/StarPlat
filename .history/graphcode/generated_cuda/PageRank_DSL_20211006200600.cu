#include "hip/hip_runtime.h"
#include"PageRank_DSL.h"

template <typename T>
__global__ void initKernel(unsigned V, T* init_array, T initVal)
{
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if(id < V)
  {
    init_array[id]=initVal;
  }
}
__global__ void Compute_PR_kernel(graph g,float beta,float delta,int maxIter,
  float* pageRank)
{
  unsigned int id = threadIdx.x + (blockDim.x * blockIdx.x);
  unsigned int v = id;
  if (id < V) 

}
void SSSP(int * OA , int * edgeList , int* cpu_edgeLen  , int src ,int V, int E )
{
  int MAX_VAL = 2147483647 ;
  int * gpu_edgeList;
  int * gpu_edgeLen;
  int * gpu_dist;
  int * gpu_OA;
  bool * gpu_modified_prev;
  bool * gpu_modified_next;
  bool * gpu_finished;


  hipMalloc(&gpu_OA,sizeof(int) *(1+V));
  hipMalloc(&gpu_edgeList,sizeof(int) *(E));
  hipMalloc(&gpu_edgeLen,sizeof(int) *(E));
  hipMalloc(&gpu_dist,sizeof(int) *(V));
  hipMalloc(&gpu_modified_prev,sizeof(bool) *(V));
  hipMalloc(&gpu_modified_next,sizeof(bool) *(V));
  hipMalloc(&gpu_finished,sizeof(bool) *(1));

  unsigned int block_size;
  unsigned int num_blocks;
   if(V <= 1024)
   {
    block_size  = V;
    block_size  = 1;
  }
  else
  {
    block_size = 1024;
    num_blocks = ceil(((float)V) / block_size);
  }
  bool *  = new bool[1];
  [0] = false;
  for (int t = 0; t < V; t ++) 
  {
    pageRank[t] = 1 / num_nodes;
  }
  hipMemcpy (gpu_OA, OA, sizeof(int) *(1+V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeList, edgeList, sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_edgeLen, cpu_edgeLen , sizeof(int) *(E) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_dist, dist, sizeof(int) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_modified_prev, modified , sizeof(bool) *(V) ,hipMemcpyHostToDevice);
  hipMemcpy (gpu_finished, finished , sizeof(bool) *(1) ,hipMemcpyHostToDevice);

  int k =0;

  while ( ![0] )
  {
    [0] =true;
    initKernel<bool><<<1,1>>>(1, gpu_finished, true);
    Compute_SSSP_kernel<<<num_blocks , block_size>>>(gpu_OA,gpu_edgeList, gpu_edgeLen ,gpu_dist,src, V ,MAX_VAL , gpu_modified_prev, gpu_modified_next, gpu_finished);
    hipDeviceSynchronize();
    initKernel<bool><<<num_blocks,block_size>>>(V, gpu_modified_prev, false);
    hipDeviceSynchronize();
    bool *tempModPtr  = gpu_modified_next;
    gpu_modified_next = gpu_modified_prev;
    gpu_modified_prev = tempModPtr;
    ++k;
    if(k==V)
    {
      break;
    }
  }

  hipMemcpy(dist,gpu_dist , sizeof(int) * (V), hipMemcpyDeviceToHost);
  char *outputfilename = "output.txt";
  FILE *outputfilepointer;
  outputfilepointer = fopen(outputfilename, "w");
  for (int i = 0; i <V; i++)
  {
    fprintf(outputfilepointer, "%d  %d\n", i, dist[i]);
  }

}
