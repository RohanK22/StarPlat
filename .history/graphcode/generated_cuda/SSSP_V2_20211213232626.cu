#include "hip/hip_runtime.h"
#include"SSSP_V2.h"

__global__ void Compute_SSSP_kernel(graph& g,int* dist,int src)

{
  bool* modified=new bool[g.num_nodes()];
  bool* modified_nxt=new bool[g.num_nodes()];
  #pragma omp parallel for  
  for (int t = 0; t < g.num_nodes(); t ++) 
  {
    dist[t] = INT_MAX;
    modified[t] = false;
    modified_nxt[t] = false;
  }
  modified[src] = true;
  dist[src] = 0;
  bool finished = false;
  while ( !finished )
  {
    finished = true;
    {
       = }
  }

}