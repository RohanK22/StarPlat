#include "color.h"


void colorGraph(graph& g){
    int V = g.num_nodes();
    int E = g.num_edges();

    printf("#nodes:%d\n",V);
    printf("#edges:%d\n",E);
    int* edgeLen = g.getEdgeLen();

    int *h_meta;
    int *h_data;
    int *h_src;
    int *h_weight;
    int *h_rev_meta;

    h_meta = (int *)malloc( (V+1)*sizeof(int));
    h_data = (int *)malloc( (E)*sizeof(int));
    h_src = (int *)malloc( (E)*sizeof(int));
    h_weight = (int *)malloc( (E)*sizeof(int));
    h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

    for(int i=0; i<= V; i++) {
        int temp = g.indexofNodes[i];
        h_meta[i] = temp;
        temp = g.rev_indexofNodes[i];
        h_rev_meta[i] = temp;
    }

    for(int i=0; i< E; i++) {
        int temp = g.edgeList[i];
        h_data[i] = temp;
        temp = g.srcList[i];
        h_src[i] = temp;
        temp = edgeLen[i];
        h_weight[i] = temp;
    }


    int* d_meta;
    int* d_data;
    int* d_src;
    int* d_weight;
    int* d_rev_meta;

    hipMalloc(&d_meta, sizeof(int)*(1+V));
    hipMalloc(&d_data, sizeof(int)*(E));
    hipMalloc(&d_src, sizeof(int)*(E));
    hipMalloc(&d_weight, sizeof(int)*(E));
    hipMalloc(&d_rev_meta, sizeof(int)*(V+1));
 
    hipMemcpy(d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
    hipMemcpy(d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
    hipMemcpy(d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, sizeof(int)*(E), hipMemcpyHostToDevice);
    hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);

    // CSR END
    //LAUNCH CONFIG
    const unsigned threadsPerBlock = 512;
    unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
    unsigned numThreads   = (V < threadsPerBlock)? V: 512;

    // TIMER START
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    unsigned int* d_color;unsigned int* h_color;
    hipMalloc(&d_color,sizeof(unsigned int)*(V+1));
    h_color = (unsigned int*)malloc(sizeof(unsigned int)*(V+1));
    bool* d_modified;
    hipMalloc(&d_modified,sizeof(bool)*(V+1));
    bool* d_modified_next;
    hipMalloc(&d_modified_next,sizeof(bool)*(V+1));

    initKernel<unsigned int> <<<numBlocks,threadsPerBlock>>>(V,d_color,(unsigned int)0);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    int fpoint1 = 0;
    int iter = 0;
    while(fpoint1<V){
        hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1),&fpoint1,sizeof(int),0,hipMemcpyHostToDevice);
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        hiprandSetPseudoRandomGeneratorSeed(gen,rand());       
        hiprandGenerate(gen,d_color,(V+1));
        compute_colors<<<numBlocks,numThreads>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_color,d_modified,d_modified_next);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&fpoint1,HIP_SYMBOL(::fpoint1),sizeof(int),0,hipMemcpyDeviceToHost);
        hipMemcpy(d_modified,d_modified_next,(V+1)*sizeof(bool),hipMemcpyDeviceToDevice);
        iter+=1;
        // fpoint1+=1;
        printf("%d\n",fpoint1);
        hiprandDestroyGenerator(gen);
    }

    printf("num colors %d\n",iter);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %.6f ms\n", milliseconds);


}



int main(int argc,char* argv[])
{
  char *file_name = argv[1];
  graph g(file_name);
  g.parseGraph();
  colorGraph(g);
  return 0;
}
