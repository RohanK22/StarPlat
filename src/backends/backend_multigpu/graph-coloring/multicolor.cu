#include "multicolor.h"

void colorGraph(graph& g){
    int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

   FILE* fptr = fopen("num_devices.txt","r"); 
  int devicecount;
   if(fptr == NULL){ 
     hipGetDeviceCount(&devicecount); 
     } 
   else{ 
     fscanf(fptr," %d ",&devicecount); 
     fclose(fptr); 
  }
  int* h_vertex_partition;
  int *h_edges;//data
  int *h_weight;
  int *h_offset;//meta
  int* h_src;
  int *h_rev_meta;
  int h_vertex_per_device;

  h_edges = (int *)malloc( (E)*sizeof(int));
  h_weight = (int *)malloc( (E)*sizeof(int));
  h_offset = (int *)malloc( (V+1)*sizeof(int));
  h_src = (int *)malloc( (E)*sizeof(int));
  h_vertex_partition = (int*)malloc((devicecount+1)*sizeof(int));
  h_rev_meta = (int *)malloc( (V+1)*sizeof(int));
  h_vertex_per_device = V/devicecount;

  for(int i=0; i<= V; i++) {
    h_offset[i] = g.indexofNodes[i];
    h_rev_meta[i] = g.rev_indexofNodes[i];
  }

  int index = 0;
  h_vertex_partition[0]=0;
  h_vertex_partition[devicecount]=V+1;
  for(int i=1;i<devicecount;i++){
    if(i<=(V%devicecount)){
       index+=(h_vertex_per_device+1);
    }
    else{
       index+=h_vertex_per_device;
    }
    h_vertex_partition[i]=index;
  }
  for(int i=0; i< E; i++){
    h_edges[i]= g.edgeList[i];
    h_src[i]=g.srcList[i];
    h_weight[i] = edgeLen[i];
  }


  int** d_offset;
  int** d_edges;
  int** d_weight;
  int** d_src;
  int** d_rev_meta;
  d_offset = (int**) malloc(devicecount*sizeof(int*));
  d_edges = (int**) malloc(devicecount*sizeof(int*));
  d_weight = (int**) malloc(devicecount*sizeof(int*));
  d_src = (int**) malloc(devicecount*sizeof(int*));
  d_rev_meta = (int**) malloc(devicecount*sizeof(int*));

  int perdevicevertices;
  int lastleftvertices;
  perdevicevertices = V / devicecount ;
  lastleftvertices = V % devicecount;
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMalloc(&d_offset[i], (V+1)*sizeof(int) );
    hipMalloc(&d_edges[i], (E)*sizeof(int) );
    hipMalloc(&d_weight[i], (E)*sizeof(int) );
    hipMalloc(&d_src[i], (E)*sizeof(int) );
    hipMalloc(&d_rev_meta[i], (V+1)*sizeof(int) );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipMemcpyAsync(d_offset[i], h_offset, (V+1)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_edges[i], h_edges, (E)*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpyAsync(d_weight[i], h_weight, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_src[i], h_src, (E)*sizeof(int),hipMemcpyHostToDevice );
    hipMemcpyAsync(d_rev_meta[i], h_rev_meta, (V+1)*sizeof(int),hipMemcpyHostToDevice );
  }
  for(int i=0;i<devicecount;i++)	
  {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }

  const unsigned threadsPerBlock = 1024;
  unsigned numThreads   = (V < threadsPerBlock)? V: 1024;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  unsigned int** d_color ;
  d_color = (unsigned int**)malloc(sizeof(unsigned int*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_color[i],sizeof(unsigned int)*(V+1));
  }


  bool** d_modified;
  d_modified = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_modified[i],sizeof(bool)*(V+1));
  }
  bool** d_modified_next;
  d_modified_next = (bool**)malloc(sizeof(bool*)*devicecount);
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    hipMalloc(&d_modified_next[i],sizeof(bool)*(V+1));
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<unsigned int><<<numBlocks,threadsPerBlock>>>(V,d_color[i],(unsigned int)0);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V,d_modified[i],(bool)false);
    hipDeviceSynchronize();
  }
  for(int i=0;i<devicecount;i++){
    hipSetDevice(i);
    initKernel<bool><<<numBlocks,threadsPerBlock>>>(V,d_modified_next[i],(bool)false);
    hipDeviceSynchronize();
  }

  int fpoint1 = 0; // asst in .cu 
  int** h_fpoint1;
  h_fpoint1 = (int**)malloc(sizeof(int*)*(devicecount+1));
  for(int i=0;i<=devicecount;i+=1){
    h_fpoint1[i] = (int*)malloc(sizeof(int));
  }

  int** d_fpoint1;
  d_fpoint1 = (int**)malloc(sizeof(int*)*devicecount);
  for(int i = 0 ; i < devicecount ; i++){
    hipSetDevice(i);
    hipMalloc(&d_fpoint1[i],sizeof(int));
    initKernel<int> <<<1,1>>>(1,d_fpoint1[i],(int)0);
  }

  int iter = 0;
  while(fpoint1<V){
    iter+=1;
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        curandCreateGenerator_t gen;
        hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
        hiprandSetPseudoRandomGeneratorSeed(gen,rand());
        hiprandGenerate(gen,d_color[i],(V+1));
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        compute_colors<<<numBlocks,numThreads>>>(h_vertex_partition[i],h_vertex_partition[i+1],V,E,d_offset[i],d_edges[i],d_src[i],d_rev_meta[i],d_color[i],d_modified[i],d_modified_next[i],d_fpoint1[i]);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i+=1){
        hipSetDevice(i);
        fpoint1+=d_fpoint1[i];
        hipMemcpyAsync(h_fpoint1[i],d_fpoint1[i],sizeof(unsigned int),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i+=1){
        hipSetDevice(i);
    }
    bool* h_modified_next;
    h_modified_next =(bool*)malloc((V+1)*sizeof(bool));
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(h_modified_next+i*(V+1),d_modified_next[i],sizeof(bool)*(V+1),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    for(int i=0;i<devicecount;i++){
        hipSetDevice(i);
        hipMemcpyAsync(d_modified[i],h_modified_next,sizeof(bool)*(V+1),hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }

  }
  printf("num colors %d\n",iter);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);
}